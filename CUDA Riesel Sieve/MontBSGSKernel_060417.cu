#include "hip/hip_runtime.h"
//Complete - Read in an ABCD file
//Complete - CUDA code with correct outputs

//TODO - Fix inputs to the CUDA function call to be an actual ABCD file and list of primes
//TODO - Function to create list of primes


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <list>

using namespace std;

hipError_t addWithCuda(int *NOut, unsigned long long *KernelP, unsigned int size, const int blocks, const int threads, unsigned long long *hashKeys, int *hashValues, int hashTableElements);


int *dev_a = 0; //NOut
unsigned long long *dev_b = 0; //KernelP
int *dev_c = 0; //kns
int *dev_e = 0; //Base
int *dev_f = 0; //counterIn
unsigned long long *dev_g = 0; //HashTable Keys
int *dev_h = 0; //HashTable Values
hipError_t cudaStatus;


__device__ unsigned long long xbinGCD(unsigned long long a, unsigned long long b)
{
	unsigned long long alpha, beta, u, v;
	u = 1; v = 0;
	alpha = a; beta = b; // Note that alpha is
						 // even and beta is odd.
						 /* The invariant maintained from here on is:
						 2a = u*2*alpha - v*beta. */
	while (a > 0) {
		a = a >> 1;
		if ((u & 1) == 0) { // Delete a common
			u = u >> 1; v = v >> 1; // factor of 2 in
		} // u and v.
		else {
			/* We want to set u = (u + beta) >> 1, but
			that can overflow, so we use Dietz's method. */
			u = ((u ^ beta) >> 1) + (u & beta);
			v = (v >> 1) + alpha;
		}
	}
	//*pu = u;
	//*pv = v;
	return v;
}

__device__ unsigned long long modul64(unsigned long long x, unsigned long long y, unsigned long long z) {
	/* Divides (x || y) by z, for 64-bit integers x, y,
	and z, giving the remainder (modulus) as the result.
	Must have x < z (to get a 64-bit result). This is
	checked for. */
	long long i, t;
	if (x >= z) {
		printf("Bad call to modul64, must have x < z.");
	}
	for (i = 1; i <= 64; i++) { // Do 64 times.
		t = (long long)x >> 63; // All 1's if x(63) = 1.
		x = (x << 1) | (y >> 63); // Shift x || y left
		y = y << 1; // one bit.
		if ((x | t) >= z) {
			x = x - z;
			y = y + 1;
		}
	}
	return x; // Quotient is y.
}

__device__ __forceinline__ unsigned long long montmul(unsigned long long abar, unsigned long long bbar, unsigned long long m, unsigned long long mprime) {
	unsigned long long thi, tlo, tm, tmmhi, tmmlo;
	unsigned long long uhi, ulo;
	unsigned int ov;
	//mulul64(abar, bbar, &thi, &tlo); // t = abar*bbar.
	thi = __umul64hi(abar, bbar);
	tlo = abar*bbar;
	/* Now compute u = (t + ((t*mprime) & mask)*m) >> 64.
	The mask is fixed at 2**64-1. Because it is a 64-bit
	quantity, it suffices to compute the low-order 64
	bits of t*mprime, which means we can ignore thi. */
	tm = tlo*mprime;
	//mulul64(tm, m, &tmmhi, &tmmlo); // tmm = tm*m.
	tmmhi = __umul64hi(tm, m);
	tmmlo = tm*m;
	
	//Replace this with ptx
	ulo = tlo + tmmlo; // Add t to tmm
	uhi = thi + tmmhi; // (128-bit add).
	if (ulo < tlo) uhi = uhi + 1; // Allow for a carry.
	// The above addition can overflow. Detect that here.
	ov = (uhi < thi) | ((uhi == thi) & (ulo < tlo));

	//asm("add.cc.u64 %0, %3, %4;\n\t" //Add tlo and tmmlo and set carry out. 
	//	"addc.cc.u64 %1, %5, %6;\n\t" //Add thi and tmmhi, use the previous carry and set carry out.
	//	"addc.u32 %2, 0, 0;" //This sets ov to 1 if the previous addition overflowed.
	//	: "=l"(ulo), "=l"(uhi) "=r"(ov) : "l"(tlo), "l"(tmmlo), "l"(thi), "l"(tmmhi)
	//	);

//	asm("add.cc.u64 %0, %0, %3;\n\t" //Add tlo and tmmlo and set carry out. 
//		"addc.cc.u64 %1, %1, %4;\n\t" //Add thi and tmmhi, use the previous carry and set carry out.
//		"addc.u32 %2, 0, 0;" //This sets ov to 1 if the previous addition overflowed.
//		: "=l"(tlo), "=l"(thi) "=r"(ov) : "l"(tmmlo), "l"(tmmhi)
//		);

//	if (ov > 0 || thi >= m) // If u >= m,
//		thi = thi - m; // subtract m from u.
//	return thi;



	ulo = uhi; // Shift u right
	if (ov > 0 || ulo >= m) // If u >= m,
		ulo = ulo - m; // subtract m from u.
	return ulo;
}

__device__ long long binExtEuclid(long long a, long long b) {
	long long u = b;
	long long v = a;
	long long r = 0;
	long long s = 1;
	long long x = a;
	while (v>0) {
		if ((u & 1) == 0) {
			u = u >> 1;
			if ((r & 1) == 0) {
				r = r >> 1;
			}
			else {
				r = (r + b) >> 1;
			}
		}
		else {
			if ((v & 1) == 0) {
				v = v >> 1;
				if ((s & 1) == 0) {
					s = s >> 1;
				}
				else {
					s = (s + b) >> 1;
				}
			}
			else {
				x = u - v;
				if (x>0) {
					u = x;
					r = r - s;
					if (r<0) {
						r = r + b;
					}
				}
				else {
					v = x * -1;
					s = s - r;
					if (s<0) {
						s = s + b;
					}
				}
			}
		}
	}
	if (r >= b) {
		r = r - b;
	}
	if (r<0) {
		r = r + b;
	}
	return(r);
}


__global__ void addKernel1(int *NOut, unsigned long long *KernelP, int *ks, int *Base, int *counterIn, unsigned long long *hashKeys, int *hashValues)
{
	clock_t beginfull = clock();
	clock_t begin = clock();
	int i = threadIdx.x;
	int block = blockIdx.x;
	int N = blockDim.x * gridDim.x; //This is threads*blocks
	
	//This deals with the hashTables
	int m = 512;
	int shift = 9; //m=2^shift
	int mem = m * 4; //This is hashTableElements * 4 to reduce collisions. Must be a power of 2
	int memN = (N * mem)-1; //We use this for doing cheap modulo's as N*mem should be a power of 2
	
	int S = block * blockDim.x; //This is this block ID*threads in a block
	unsigned long long b = KernelP[S + i];
	//my_uint64_t KernelBase;
	//KernelBase.y = *Base;
	int KernelBase = *Base;
	int outputBase = KernelBase;
	int counter = *counterIn;
	int barretts = 0;

	bool printer = false;
	if (i == 0 & block == 0) {
		printer = true;
	}

	clock_t end = clock();
	int time_spent = (end - begin);
	if (printer) {
		printf("Cycles to complete variable setup was %d\n", time_spent);
	}

	begin = clock();
	//Calculate m = floor(2^128/b) using div_128_64. 
	//my_uint128_t m1;
	//int128_t m1 = div_128_64(b);

	unsigned long long bprime = xbinGCD(9223372036854775808, b);

	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles to do xbinGCD was %d\n", time_spent);
	}

	begin = clock();
	//Try to populate our new hash table array ------------------------------------------------------------
	int lookups = 0;
	int hash = 0;
	//hashTable[1].key = 1;
	unsigned long long js = 1;
	//my_uint64_t js;
	//js.y = 1;

	//Convert js to montgomery space
	js = modul64(js, 0, b);
	KernelBase = modul64(KernelBase, 0, b);

	//Keep the js values in a local array
	//unsigned long long jsLocal[512];

	for (int j = 0; j<m; j++) {

		//jsLocal[j] = js;

		hash = js & mem-1;
		//hash = js.y & 4095;
		int index = hash*N + S + i;

		//Basic linear probing
		for (int probe = 0; probe < m; probe++) {
			lookups++;
				
			if ((hashKeys[index]) == 0) {
				if (printer & j == m >> 1) {
					begin = clock();
				}
				hashKeys[index] = js;
				//hashKeys[index] = js.y;
				hashValues[index] = j;
				if (printer & j == m >> 1) {
					end = clock();
					time_spent = (end - begin);
					printf("Cycles to add key and value to hash table was %d\n", time_spent);
				}
				break;
			}

			//if (hashValues[index] == 0) {
			//	if (printer & j == m >> 1) {
			//		begin = clock();
			//	}
			//	hashValues[index] = j;
			//	if (printer & j == m >> 1) {
			//		end = clock();
			//		time_spent = (end - begin);
			//		printf("Cycles to add key and value to hash table was %d\n", time_spent);
			//	}
			//	break;
			//}

			index = index + N;
			index = index & memN;
		}

		if (printer & j == m >> 1) {
			begin = clock();
		}
		//js = barrett(m1, js, KernelBase, b);
		js = montmul(js, KernelBase, b, bprime);
		barretts++;
		if (printer & j == m >> 1) {
			end = clock();
			time_spent = (end - begin);
			printf("Cycles to do a montgomery multiply was %d\n", time_spent);
		}
	}
	if (printer) {
		printf("Number of lookups while inserting into the hash table was %d\n", lookups);
	}

	//Finished calculating the hash table --------------------------------------------------------------------

	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles calculating new hash table was %d\n", time_spent);
		printf("Average was %d\n", time_spent/m);
	}


	begin = clock();
	//Compute KernelBase^-m (mod b)
	unsigned long long c1 = binExtEuclid(outputBase, b);
	//unsigned long long int c1Old = binExtEuclid(KernelBase.y, b);

	c1 = modul64(c1, 0, b);

	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles performing binExtEuclid was %d\n", time_spent);
	}
	
	//This should be KernelBase^-1 (mod b)
	//Now repeatedly square it as m is a power of two

	begin = clock();
	
	//my_uint64_t c1;
	//c1.x = (int)(c1Old >> 32);
	//c1.y = (int)(c1Old);
															 
	for (int t = 0; t<shift; t++) {
		//c1 = barrett(m1, c1, c1, b);
		c1 = montmul(c1, c1, b, bprime);
		barretts++;
	}

	long long output = -5;
	int NMin = 20;
	int NMax = 1000;
	int tMin = NMin >> shift;
	lookups = 0;
	int countmuls = tMin;

	for (int k = 0; k<counter; k++) {
		//So work out beta from it
		unsigned long long beta = binExtEuclid(ks[k], b);
		beta = modul64(beta, 0, b);
		//unsigned long long betaOld = binExtEuclid(ks[k], b);
		//my_uint64_t beta;
		//beta.x = (int)(betaOld >> 32);
		//beta.y = (int)(betaOld);
		
		for (int t = 0; t<tMin; t++) {
			//beta = barrett(m1, beta, c1, b);
			beta = montmul(beta, c1, b, bprime);
			barretts++;
		}

		for (int t = tMin; t<140; t++) {

			//Check if beta is in js
			int hash = beta & mem-1;
			//int hash = beta.y & 4095;
			int index = hash*N + S + i;

			//Its possible beta is here, use linear probing to check
			for (int probe = 0; probe < m; probe++) {
				lookups++;
				//int localIndex = hashValues[index];
				//unsigned long long localKey = hashKeys[index];
				if (hashKeys[index] == 0) {
				//if (localIndex == 0) {
					//Beta is not here
					break;
				}
				else if (hashKeys[index]  == beta) {
				//else if (jsLocal[localIndex] == beta) {
				//else if ((hashKeys[index]) == beta.y) {
					lookups++;
					//We've found beta
					//We've had a match
					//output = (t*m + (localIndex));
					output = (t*m + (hashValues[index]));
					//printf("Match via hash with probing in Thread %d, Block %d. t=%d, hash=%d, probe=%d beta=%llu. Output will be %llu | %d*%d^%d-1\n", i, block, t, hash, probe, beta, b, ks[k], outputBase, output);
					break;
				}
				index = index + N;
				index = index & memN;
			}
			
			//beta = barrett(m1, beta, c1, b);
			beta = montmul(beta, c1, b, bprime);
			countmuls++;
			barretts++;
		}

		if (output < NMin) {
			output=-3;
		}
		else if (output > NMax) {
			output = -4;
		}
		else {
			printf("Output will be %llu | %d*%d^%d-1\n", b, ks[k], outputBase, output);

		}
	}
	if (printer) {
		printf("Number of lookups against hash table was %d\n", lookups);
	}

	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles to complete BSGS step was %d\n", time_spent);
		printf("Average (BSGS Cycles/muls) was %d\n", (time_spent/countmuls));
		printf("Average (BSGS Cycles/lookups) was %d\n", (time_spent / lookups));
	}

	begin = clock();

	NOut[S + i] = output; //This contains the k-value in the top 32 bits and the n-value in the low 32 bits

	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles to write output to NOut was %d\n", time_spent);
	}

	if (printer) {
		printf("Total number of montgomery multiplies was %d\n", barretts);
	}

	time_spent = (end - beginfull);
	if (printer) {
		printf("Cycles to execute one full thread was %d\n", time_spent);
	}
}



int main()
{

	//Read in an ABCD file and parse ----------------------------------------------------------------------------
	string line;
	int total = 0;
	//string abcdFile = "C:\\Users\\Rob\\Documents\\Visual Studio 2015\\Projects\\CPU Sieve\\sr_108.abcd";
	//string abcdFile = "C:\\Users\\Rob\\Desktop\\TestSieve\\sr_745.abcd";
	string abcdFile = "sr_745.abcd";

	//First pass through the ABCD file to find the number of k's and max number of n's
	int count1 = 0; //Number of k's
	int count3 = 0; //Total number of lines
	ifstream myfile(abcdFile);
	if (myfile.is_open())
	{
		while (getline(myfile, line))
		{
			count3++;

			string::size_type n = line.find(" ");
			string token = line.substr(0,n);
			//cout << token << endl;

			//If tokens[0] == "ABCD" then this defines a new k, otherwise it is a number
			if (token.compare("ABCD") == 0) {
				count1++;
				//cout << "We're here!" << endl;
			}
		}
		myfile.close();
	}

	else cout << "Unable to open file first time" << endl;

	//Second pass through the ABCD file to write the values into the matrix
	//Store the k and n values in this array
	//boost::numeric::ublas::matrix<int> kns(count1, max);
	//std::list<int> kns;
	count3 = count3 + (2 * count1);
	int *kns = (int *)malloc(count3*sizeof(int));
	int *ks = (int *)malloc(count1*sizeof(int));

	//Reset the counts
	count1 = 0;
	count3 = 0;
	int base = 0;
	ifstream myfile2(abcdFile);
	if (myfile2.is_open())
	{
		while (getline(myfile2, line))
		{
			//Tokenise the string - if the first element of the string is "ABCD" then this is a new k-value
			string::size_type n = line.find(" ");
			string token = line.substr(0, n);
			//If tokens[0] == "ABCD" then this defines a new k, otherwise it is a number
			if (token.compare("ABCD") == 0) {
				cout << "We've found ABCD. Get the k-value" << endl;
				//Insert a 0 into kns before we insert the k-value
				kns[count3] = 0;
				count3++;
				//Get the k value
				token = line.substr(n+1);
				//cout << token << endl;
				n = token.find("*");
				string tok = token.substr(0, n);
				//cout << tok << endl;
				
				int kval = stoi(tok);
				kns[count3] = kval;
				count3++;
				ks[count1] = kval;
				count1++;
				//Get the base
				if (base == 0) {
					token = token.substr(n+1);
					n = token.find("^");
					string b = token.substr(0, n);
					//cout << b << endl;
					base = stoi(b);
					cout << "The base is " << base << endl;
				}
				//Get the starting n-value - remove the square brakets
				n = token.find("[");
				token = token.substr(n + 1);
				n = token.find("]");
				token = token.substr(0, n);
				//cout << token << endl;
				total = stoi(token);
				kns[count3] = total;
				count3++;
				cout << "This is a new k-value with value " << kval << " and initial n-value " << total << endl;
			}
			else {
				//This is a number, n-value offset
				//cout << token << endl;
				int offset = stoi(token);
				total = total + offset;
				kns[count3] = total;
				count3++;
				//cout << count3 << endl;
			}

		}

		myfile2.close();
	}

	else cout << "Unable to open file second time" << endl;

	//End of reading ABCD file ----------------------------------------------------------------------------------
	cout << "End of reading ABCD file" << endl;


	//Generate Primes -------------------------------------------------------------------------------------------

	const int blocks = 256; 
	const int threads = 128; //These must multiply to around 65536. Larger and CUDA times out
    const int arraySize = blocks*threads;
	const int testArraySize = arraySize * 24;
	const int hashTableElements = 512;
	unsigned long long *KernelP = (unsigned long long *)malloc(arraySize*sizeof(unsigned long long));
    //unsigned long long KernelP[arraySize] = { 0 };
	int *NOut = (int *)malloc(arraySize*sizeof(int));
    //int NOut[arraySize] = { 0 };
	unsigned long long *hashKeys = (unsigned long long *)malloc(arraySize * hashTableElements * 4 * sizeof(unsigned long long));
	memset(hashKeys, 0, arraySize * hashTableElements * 4 * sizeof(unsigned long long));
	int *hashValues = (int *)malloc(arraySize * hashTableElements * 4 * sizeof(int));
	memset(hashValues, 0, arraySize * hashTableElements * 4 * sizeof(int));

	//Low should be greater than the primes we use below. 
	unsigned long long low = 6000000000;
	unsigned long long high = 6003000000;

	//unsigned long long low = 600000;
	//unsigned long long high = 7000000;

	unsigned long long startLow = low; //Don't touch this. Used for timing purposes

	//Use the idea of a segmented sieve. Generate a list of small primes first
	//Could use the first 1024 primes as a starter. 8161 is the 1024th prime
	//Currently using the first 70 primes aas a starter.
	clock_t begin = clock();
	int smallPrimes = 8162;
	int primeCount = 1024;
	int s = 0;
	bool *primes = (bool *)malloc(smallPrimes*sizeof(bool));
	unsigned int *smallP = (unsigned int *)malloc(primeCount*sizeof(unsigned int));
	memset(primes, true, smallPrimes*sizeof(bool));

	int sq = smallPrimes*smallPrimes;

	for (int p = 2; p*p < sq; p++) {
		if (primes[p] == true) {
			smallP[s] = p;
			//cout << smallP[s] << endl;
			s++;
			for (int i = p*2; i < smallPrimes; i += p) {
				primes[i] = false;
			}
		}
	}
	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	cout << "Time generating small primes " << time_spent << "s" << endl;

	//Print the small primes as a check
	for (int p = 0; p < primeCount; p++) {
		//cout << smallP[p] << endl;
	}

	//Find the minimum number in [low...high] that is a multiple of primes[i]
	
	bool *mark = (bool *)malloc(testArraySize*sizeof(bool));



	//Try setting up the GPU just once

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (one output, seven input). 
	//Give all vectors same size for now, we can change this afterwards

	cudaStatus = hipMalloc((void**)&dev_a, arraySize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, arraySize * sizeof(unsigned long long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_c, count1 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_e, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_f, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_g, arraySize * hashTableElements * 4 * sizeof(unsigned long long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_h, arraySize * hashTableElements * 4 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_c, ks, count1 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_e, &base, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_f, &count1, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_g, hashKeys, arraySize * hashTableElements * 4 * sizeof(unsigned long long), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy input failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_h, hashValues, arraySize * hashTableElements * 4 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy input failed!");
		goto Error;
	}

	int kernelCount = 0;
	clock_t loopTime = clock();
	//From here we need to loop to keep the GPU busy. 
	while (low < high) {
		kernelCount++;
		cout << "Executing kernel number " << kernelCount << endl;
		cout << "Low is now set to " << low << endl;

		begin = clock();
		memset(mark, true, testArraySize*sizeof(bool));

		for (int i = 0; i < primeCount; i++) {
			unsigned int smallPrime = smallP[i];
			for (int j = 0; j < testArraySize; j++) {
				//if (mark[j] == true && (((low + j) % smallP[i]) == 0)) {
				if (((low + j) % smallPrime) == 0) {
					//So if low + offset can be divided by i we've found the first value divisible by i. Now mark off all i multiples
					for (int k = j; k < testArraySize; k += smallPrime) {
						mark[k] = false;
					}
					break;
				}
			}
		}
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		cout << "Time marking the prime array " << time_spent << "s" << endl;


		// Numbers which are not marked as false are prime
		begin = clock();
		int countPrimes = 0;
		for (unsigned long long i = low; i < low + (testArraySize); i++) {
			if (mark[i - low] == true) {
				KernelP[countPrimes] = i;
				countPrimes++;
				if (countPrimes == arraySize) {
					cout << "We got as far as " << i << " out of " << low + (testArraySize) << endl;
					break;
				}
				//cout << i << endl;
			}
		}
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		cout << "Time generating kernel primes " << time_spent << "s" << endl;

		unsigned long long minPrime = KernelP[0];
		unsigned long long maxPrime = KernelP[arraySize - 1];
		unsigned long long progress = maxPrime - minPrime;

		cout << "Min Prime = " << minPrime << ". Max Prime = " << maxPrime << ". Progress = " << progress << endl;
		cout << "Array Size = " << arraySize << endl;

		//End of Generating Primes ----------------------------------------------------------------------------------

		begin = clock();
		cout << "Try to launch the CUDA kernel" << endl;
		// Add vectors in parallel.
		//This uses the full ABCD file, but runs very slowly when file is big
		//hipError_t cudaStatus = addWithCuda(NOut, KernelP, kns, &base, &count3, arraySize, count3, blocks, threads);
		//This is datless - remember to change to addkernel1
		hipError_t cudaStatus = addWithCuda(NOut, KernelP, arraySize, blocks, threads, hashKeys, hashValues, hashTableElements);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		cout << "Time to execute kernel (outside function) " << time_spent << "s" << endl;

		printf("%d,%d,%d,%d,%d,%d,%d,%d,%d,%d\n\n",
			NOut[0], NOut[1], NOut[2], NOut[3], NOut[4], NOut[5], NOut[6], NOut[7], NOut[8], NOut[9]);

		//Set low equal to high and continue in the loop
		low = maxPrime;
	}

	clock_t loopEnd = clock();
	time_spent = (double)(loopEnd - loopTime) / CLOCKS_PER_SEC;
	cout << "Time taken " << time_spent << "s" << endl;
	cout << "Time per kernel " << time_spent / kernelCount << endl;
	cout << "Progress = " << KernelP[arraySize - 1] - startLow << " at " << (KernelP[arraySize - 1] - startLow) / time_spent << " p/sec" << endl;

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_e);
	hipFree(dev_f);
	hipFree(dev_g);
	hipFree(dev_h);
	return cudaStatus;

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *NOut, unsigned long long *KernelP, unsigned int size, const int blocks, const int threads, unsigned long long *hashKeys, int *hashValues, int hashTableElements)
{

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_b, KernelP, size * sizeof(unsigned long long), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy input failed!");
    }

	cudaStatus = hipMemset(dev_g, 0, size * hashTableElements * 4 * sizeof(unsigned long long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy input failed!");
	}

	//cudaStatus = hipMemset(dev_h, 0, size * hashTableElements * 4 * sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy input failed!");
	//}


	//hipEvent_t start, stop;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);
    // Launch a kernel on the GPU with one thread for each element.
	//hipEventRecord(start);
	hipStream_t stream0;
	//hipStream_t stream1;
	hipStreamCreate(&stream0);
	//hipStreamCreate(&stream1);
    addKernel1<<<blocks,threads,0,stream0>>>(dev_a, dev_b, dev_c, dev_e, dev_f, dev_g, dev_h);
	//addKernel1<<<blocks,threads,0,stream1>>>(dev_a, dev_b, dev_c, dev_e, dev_f, dev_g, dev_h);
	//hipEventRecord(stop);

	//hipEventSynchronize(stop);
	//float milliseconds = 0;
	//hipEventElapsedTime(&milliseconds, start, stop);
	//printf("Time taken: %f ms \n", milliseconds);

    // Check for any errors launching the kernel
    //cudaStatus = hipGetLastError();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    //}
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(NOut, dev_a, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy output failed!");
    }
    
    return cudaStatus;
}
