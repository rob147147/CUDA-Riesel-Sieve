#include "hip/hip_runtime.h"
//Complete - Read in an ABCD file
//Complete - CUDA code with correct outputs

//TODO - Fix inputs to the CUDA function call to be an actual ABCD file and list of primes
//TODO - Function to create list of primes


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <list>

using namespace std;

hipError_t addWithCuda(int *NOut, unsigned long long *KernelP, unsigned int size, const int blocks, const int threads, unsigned long long *hashKeys, int *hashValues, int hashTableElements);


int *dev_a = 0; //NOut
unsigned long long *dev_b = 0; //KernelP
int *dev_c = 0; //kns
int *dev_e = 0; //Base
int *dev_f = 0; //counterIn
unsigned long long *dev_g = 0; //HashTable Keys
int *dev_h = 0; //HashTable Values
hipError_t cudaStatus;


__device__ unsigned long long xbinGCD(unsigned long long a, unsigned long long b)
{
	unsigned long long alpha, beta, u, v;
	u = 1; v = 0;
	alpha = a; beta = b; // Note that alpha is
						 // even and beta is odd.
						 /* The invariant maintained from here on is:
						 2a = u*2*alpha - v*beta. */
	while (a > 0) {
		a = a >> 1;
		if ((u & 1) == 0) { // Delete a common
			u = u >> 1; v = v >> 1; // factor of 2 in
		} // u and v.
		else {
			/* We want to set u = (u + beta) >> 1, but
			that can overflow, so we use Dietz's method. */
			u = ((u ^ beta) >> 1) + (u & beta);
			v = (v >> 1) + alpha;
		}
	}
	//*pu = u;
	//*pv = v;
	return v;
}

__device__ unsigned long long modul64(unsigned long long x, unsigned long long y, unsigned long long z) {
	/* Divides (x || y) by z, for 64-bit integers x, y,
	and z, giving the remainder (modulus) as the result.
	Must have x < z (to get a 64-bit result). This is
	checked for. */
	long long i, t;
	if (x >= z) {
		printf("Bad call to modul64, must have x < z.");
	}
	for (i = 1; i <= 64; i++) { // Do 64 times.
		t = (long long)x >> 63; // All 1's if x(63) = 1.
		x = (x << 1) | (y >> 63); // Shift x || y left
		y = y << 1; // one bit.
		if ((x | t) >= z) {
			x = x - z;
			y = y + 1;
		}
	}
	return x; // Quotient is y.
}

__device__ __forceinline__ unsigned long long montmul(unsigned long long abar, unsigned long long bbar, unsigned long long m, unsigned long long mprime) {
	unsigned long long thi, tlo, tm, tmmhi, tmmlo;
	unsigned long long uhi, ulo;
	unsigned int ov;
	//mulul64(abar, bbar, &thi, &tlo); // t = abar*bbar.
	thi = __umul64hi(abar, bbar);
	tlo = abar*bbar;
	/* Now compute u = (t + ((t*mprime) & mask)*m) >> 64.
	The mask is fixed at 2**64-1. Because it is a 64-bit
	quantity, it suffices to compute the low-order 64
	bits of t*mprime, which means we can ignore thi. */
	tm = tlo*mprime;
	//mulul64(tm, m, &tmmhi, &tmmlo); // tmm = tm*m.
	tmmhi = __umul64hi(tm, m);
	tmmlo = tm*m;
	
	//Replace this with ptx
	//ulo = tlo + tmmlo; // Add t to tmm
	//uhi = thi + tmmhi; // (128-bit add).
	//if (ulo < tlo) uhi = uhi + 1; // Allow for a carry.
	// The above addition can overflow. Detect that here.
	//ov = (uhi < thi) | ((uhi == thi) & (ulo < tlo));

	asm("add.cc.u64 %0, %3, %4;\n\t" //Add tlo and tmmlo and set carry out. 
		"addc.cc.u64 %1, %5, %6;\n\t" //Add thi and tmmhi, use the previous carry and set carry out.
		"addc.u32 %2, 0, 0;" //This sets ov to 1 if the previous addition overflowed.
		: "=l"(ulo), "=l"(uhi) "=r"(ov) : "l"(tlo), "l"(tmmlo), "l"(thi), "l"(tmmhi)
		);

//	asm("add.cc.u64 %0, %0, %3;\n\t" //Add tlo and tmmlo and set carry out. 
//		"addc.cc.u64 %1, %1, %4;\n\t" //Add thi and tmmhi, use the previous carry and set carry out.
//		"addc.u32 %2, 0, 0;" //This sets ov to 1 if the previous addition overflowed.
//		: "=l"(tlo), "=l"(thi) "=r"(ov) : "l"(tmmlo), "l"(tmmhi)
//		);

//	if (ov > 0 || thi >= m) // If u >= m,
//		thi = thi - m; // subtract m from u.
//	return thi;



	ulo = uhi; // Shift u right
	if (ov > 0 || ulo >= m) // If u >= m,
		ulo = ulo - m; // subtract m from u.
	return ulo;
}

__device__ long long binExtEuclid(long long a, long long b) {
	long long u = b;
	long long v = a;
	long long r = 0;
	long long s = 1;
	long long x = a;
	while (v>0) {
		if ((u & 1) == 0) {
			u = u >> 1;
			if ((r & 1) == 0) {
				r = r >> 1;
			}
			else {
				r = (r + b) >> 1;
			}
		}
		else {
			if ((v & 1) == 0) {
				v = v >> 1;
				if ((s & 1) == 0) {
					s = s >> 1;
				}
				else {
					s = (s + b) >> 1;
				}
			}
			else {
				x = u - v;
				if (x>0) {
					u = x;
					r = r - s;
					if (r<0) {
						r = r + b;
					}
				}
				else {
					v = x * -1;
					s = s - r;
					if (s<0) {
						s = s + b;
					}
				}
			}
		}
	}
	if (r >= b) {
		r = r - b;
	}
	if (r<0) {
		r = r + b;
	}
	return(r);
}


__global__ void addKernel1(int *NOut, unsigned long long *KernelP, int *ks, int *Base, int *counterIn, unsigned long long *hashKeys, int *hashValues)
{
	
	int i = threadIdx.x;
	int block = blockIdx.x;
	int S = block * blockDim.x; //This is this block ID*threads in a block
	unsigned long long b = KernelP[S + i];
	unsigned long long bprime = xbinGCD(9223372036854775808, b);
	int KernelBase = *Base;
	int counter = *counterIn;
	int outputBase = KernelBase;

	int loop = 500; //This should be roughly sqrt(NMax - NMin)

	long output = -5;
	int NMax = 500000;
	int NMin = 250000;
	int d = NMax;

	//Move b1 to montgomery space
	unsigned long long b1 = modul64(KernelBase, 0, b);
	//These are all montgomery mults
	unsigned long long b2 = montmul(b1, b1, b, bprime);
	unsigned long long b3 = montmul(b2, b2, b, bprime);
	unsigned long long b4 = montmul(b3, b3, b, bprime);
	unsigned long long b5 = montmul(b4, b4, b, bprime);
	unsigned long long b6 = montmul(b5, b5, b, bprime);
	unsigned long long b7 = montmul(b6, b6, b, bprime);
	unsigned long long b8 = montmul(b7, b7, b, bprime);
	unsigned long long b9 = montmul(b8, b8, b, bprime);
	unsigned long long b10 = montmul(b9, b9, b, bprime);
	unsigned long long b11 = montmul(b10, b10, b, bprime);
	unsigned long long b12 = montmul(b11, b11, b, bprime);
	unsigned long long b13 = montmul(b12, b12, b, bprime);
	//ulong b14 = montmul(b13,b13,b,rInvMdash);
	//ulong b15 = montmul(b14,b14,b,rInvMdash);
	//ulong b16 = montmul(b15,b15,b,rInvMdash);

	//Move the x0 starting point to montgomery space
	unsigned long long x0 = modul64(1, 0, b);
	int tempNMax = NMax;

	unsigned long long bInc = b1;
	int k = 32 - __clz(tempNMax);
	for (int i = 0; i<k; i++) {
		if ((tempNMax & 1) == 1) {
			x0 = montmul(x0, bInc, b, bprime);
		}
		tempNMax >>= 1;
		bInc = montmul(bInc, bInc, b, bprime);
	}

	int j = 0;
	int j1 = 0;
	unsigned long long bsnew = 0;
	int distances[4] = { 1,32,256,512 }; //The mean of these should be roughly equal to loops/2
	unsigned long long bs[4] = { b1, b3, b9, b10};

	for (int i = 0; i<loop; i++) {
		j = ((x0)& 3);
		j1 = j & 1;
		bsnew = j1 == 0 ? j == 0 ? b1 : b9 : j == 1 ? b6 : b10;
		//d = d + (1 << j << j);
		x0 = montmul(x0, bsnew, b, bprime);
		//j = (x0) & 3;
		d = d + distances[j];
		//x0 = montmul(x0, bs[j], b, bprime);
		//printf("d is up to %d\n", d);
	}

	//The loop above is only run once as it doesn't depend on c1, and the loop below is run for each c1 (each k value)

	int permD = d;
	unsigned long long c1 = 0;

	for (int c = 0; c<counter; c++) {
		d = permD;
		bool xor = 1;
		output = -5;
		c1 = binExtEuclid(ks[c], b);
		//Move this to montgomery space
		c1 = modul64(c1, 0, b);
		while (xor) {
			j = ((c1)& 3);
			j1 = j & 1;
			bsnew = j1 == 0 ? j == 0 ? b1 : b9 : j == 1 ? b6 : b10;
			//d = d - (1 << j << j);
			c1 = montmul(c1, bsnew, b, bprime);
			//j = (x0) & 3;
			d = d - distances[j];
			//c1 = montmul(c1, bs[j], b, bprime);
			//printf("d is down to $d\n", d);

			xor = (c1 != x0);
			output = d;
			if (d<NMin) {
				xor = 0;
			}

		}
		if (output < NMin) {
			output = -3;
		}
		else if (output > NMax) {
			output = -4;
		}
		else {
			printf("Output will be %llu | %d*%d^%d-1\n", b, ks[c], outputBase, output);
		}

	}

	NOut[S + i] = output;
}



int main()
{

	//Read in an ABCD file and parse ----------------------------------------------------------------------------
	string line;
	int total = 0;
	//string abcdFile = "C:\\Users\\Rob\\Documents\\Visual Studio 2015\\Projects\\CPU Sieve\\sr_108.abcd";
	//string abcdFile = "C:\\Users\\Rob\\Desktop\\TestSieve\\sr_745.abcd";
	string abcdFile = "sr_745.abcd";

	//First pass through the ABCD file to find the number of k's and max number of n's
	int count1 = 0; //Number of k's
	int count3 = 0; //Total number of lines
	ifstream myfile(abcdFile);
	if (myfile.is_open())
	{
		while (getline(myfile, line))
		{
			count3++;

			string::size_type n = line.find(" ");
			string token = line.substr(0,n);
			//cout << token << endl;

			//If tokens[0] == "ABCD" then this defines a new k, otherwise it is a number
			if (token.compare("ABCD") == 0) {
				count1++;
				//cout << "We're here!" << endl;
			}
		}
		myfile.close();
	}

	else cout << "Unable to open file first time" << endl;

	//Second pass through the ABCD file to write the values into the matrix
	//Store the k and n values in this array
	//boost::numeric::ublas::matrix<int> kns(count1, max);
	//std::list<int> kns;
	count3 = count3 + (2 * count1);
	int *kns = (int *)malloc(count3*sizeof(int));
	int *ks = (int *)malloc(count1*sizeof(int));

	//Reset the counts
	count1 = 0;
	count3 = 0;
	int base = 0;
	ifstream myfile2(abcdFile);
	if (myfile2.is_open())
	{
		while (getline(myfile2, line))
		{
			//Tokenise the string - if the first element of the string is "ABCD" then this is a new k-value
			string::size_type n = line.find(" ");
			string token = line.substr(0, n);
			//If tokens[0] == "ABCD" then this defines a new k, otherwise it is a number
			if (token.compare("ABCD") == 0) {
				cout << "We've found ABCD. Get the k-value" << endl;
				//Insert a 0 into kns before we insert the k-value
				kns[count3] = 0;
				count3++;
				//Get the k value
				token = line.substr(n+1);
				//cout << token << endl;
				n = token.find("*");
				string tok = token.substr(0, n);
				//cout << tok << endl;
				
				int kval = stoi(tok);
				kns[count3] = kval;
				count3++;
				ks[count1] = kval;
				count1++;
				//Get the base
				if (base == 0) {
					token = token.substr(n+1);
					n = token.find("^");
					string b = token.substr(0, n);
					//cout << b << endl;
					base = stoi(b);
					cout << "The base is " << base << endl;
				}
				//Get the starting n-value - remove the square brakets
				n = token.find("[");
				token = token.substr(n + 1);
				n = token.find("]");
				token = token.substr(0, n);
				//cout << token << endl;
				total = stoi(token);
				kns[count3] = total;
				count3++;
				cout << "This is a new k-value with value " << kval << " and initial n-value " << total << endl;
			}
			else {
				//This is a number, n-value offset
				//cout << token << endl;
				int offset = stoi(token);
				total = total + offset;
				kns[count3] = total;
				count3++;
				//cout << count3 << endl;
			}

		}

		myfile2.close();
	}

	else cout << "Unable to open file second time" << endl;

	//End of reading ABCD file ----------------------------------------------------------------------------------
	cout << "End of reading ABCD file" << endl;


	//Generate Primes -------------------------------------------------------------------------------------------

	const int blocks = 256; 
	const int threads = 128; //These must multiply to around 65536. Larger and CUDA times out
    const int arraySize = blocks*threads;
	const int testArraySize = arraySize * 24;
	const int hashTableElements = 512;
	unsigned long long *KernelP = (unsigned long long *)malloc(arraySize*sizeof(unsigned long long));
    //unsigned long long KernelP[arraySize] = { 0 };
	int *NOut = (int *)malloc(arraySize*sizeof(int));
    //int NOut[arraySize] = { 0 };
	unsigned long long *hashKeys = (unsigned long long *)malloc(arraySize * hashTableElements * 4 * sizeof(unsigned long long));
	memset(hashKeys, 0, arraySize * hashTableElements * 4 * sizeof(unsigned long long));
	int *hashValues = (int *)malloc(arraySize * hashTableElements * 4 * sizeof(int));
	memset(hashValues, 0, arraySize * hashTableElements * 4 * sizeof(int));

	//Low should be greater than the primes we use below. 
	//unsigned long long low = 6000000000;
	//unsigned long long high = 6003000000;

	unsigned long long low = 1000067000000;
	unsigned long long high = 1000070000000;

	//unsigned long long low = 600000;
	//unsigned long long high = 7000000;

	unsigned long long startLow = low; //Don't touch this. Used for timing purposes

	//Use the idea of a segmented sieve. Generate a list of small primes first
	//Could use the first 1024 primes as a starter. 8161 is the 1024th prime
	//Currently using the first 70 primes aas a starter.
	clock_t begin = clock();
	int smallPrimes = 8162;
	int primeCount = 1024;
	int s = 0;
	bool *primes = (bool *)malloc(smallPrimes*sizeof(bool));
	unsigned int *smallP = (unsigned int *)malloc(primeCount*sizeof(unsigned int));
	memset(primes, true, smallPrimes*sizeof(bool));

	int sq = smallPrimes*smallPrimes;

	for (int p = 2; p*p < sq; p++) {
		if (primes[p] == true) {
			smallP[s] = p;
			//cout << smallP[s] << endl;
			s++;
			for (int i = p*2; i < smallPrimes; i += p) {
				primes[i] = false;
			}
		}
	}
	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	cout << "Time generating small primes " << time_spent << "s" << endl;

	//Print the small primes as a check
	for (int p = 0; p < primeCount; p++) {
		//cout << smallP[p] << endl;
	}

	//Find the minimum number in [low...high] that is a multiple of primes[i]
	
	bool *mark = (bool *)malloc(testArraySize*sizeof(bool));



	//Try setting up the GPU just once

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (one output, seven input). 
	//Give all vectors same size for now, we can change this afterwards

	cudaStatus = hipMalloc((void**)&dev_a, arraySize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, arraySize * sizeof(unsigned long long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_c, count1 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_e, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_f, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_g, arraySize * hashTableElements * 4 * sizeof(unsigned long long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_h, arraySize * hashTableElements * 4 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_c, ks, count1 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_e, &base, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_f, &count1, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_g, hashKeys, arraySize * hashTableElements * 4 * sizeof(unsigned long long), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy input failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_h, hashValues, arraySize * hashTableElements * 4 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy input failed!");
		goto Error;
	}

	int kernelCount = 0;
	clock_t loopTime = clock();
	//From here we need to loop to keep the GPU busy. 
	while (low < high) {
		kernelCount++;
		cout << "Executing kernel number " << kernelCount << endl;
		cout << "Low is now set to " << low << endl;

		begin = clock();
		memset(mark, true, testArraySize*sizeof(bool));

		for (int i = 0; i < primeCount; i++) {
			unsigned int smallPrime = smallP[i];
			for (int j = 0; j < testArraySize; j++) {
				//if (mark[j] == true && (((low + j) % smallP[i]) == 0)) {
				if (((low + j) % smallPrime) == 0) {
					//So if low + offset can be divided by i we've found the first value divisible by i. Now mark off all i multiples
					for (int k = j; k < testArraySize; k += smallPrime) {
						mark[k] = false;
					}
					break;
				}
			}
		}
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		cout << "Time marking the prime array " << time_spent << "s" << endl;


		// Numbers which are not marked as false are prime
		begin = clock();
		int countPrimes = 0;
		for (unsigned long long i = low; i < low + (testArraySize); i++) {
			if (mark[i - low] == true) {
				KernelP[countPrimes] = i;
				countPrimes++;
				if (countPrimes == arraySize) {
					cout << "We got as far as " << i << " out of " << low + (testArraySize) << endl;
					break;
				}
				//cout << i << endl;
			}
		}
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		cout << "Time generating kernel primes " << time_spent << "s" << endl;

		unsigned long long minPrime = KernelP[0];
		unsigned long long maxPrime = KernelP[arraySize - 1];
		unsigned long long progress = maxPrime - minPrime;

		cout << "Min Prime = " << minPrime << ". Max Prime = " << maxPrime << ". Progress = " << progress << endl;
		cout << "Array Size = " << arraySize << endl;

		//End of Generating Primes ----------------------------------------------------------------------------------

		begin = clock();
		cout << "Try to launch the CUDA kernel" << endl;
		// Add vectors in parallel.
		//This uses the full ABCD file, but runs very slowly when file is big
		//hipError_t cudaStatus = addWithCuda(NOut, KernelP, kns, &base, &count3, arraySize, count3, blocks, threads);
		//This is datless - remember to change to addkernel1
		hipError_t cudaStatus = addWithCuda(NOut, KernelP, arraySize, blocks, threads, hashKeys, hashValues, hashTableElements);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		cout << "Time to execute kernel (outside function) " << time_spent << "s" << endl;

		printf("%d,%d,%d,%d,%d,%d,%d,%d,%d,%d\n\n",
			NOut[0], NOut[1], NOut[2], NOut[3], NOut[4], NOut[5], NOut[6], NOut[7], NOut[8], NOut[9]);

		//Set low equal to high and continue in the loop
		low = maxPrime;
	}

	clock_t loopEnd = clock();
	time_spent = (double)(loopEnd - loopTime) / CLOCKS_PER_SEC;
	cout << "Time taken " << time_spent << "s" << endl;
	cout << "Time per kernel " << time_spent / kernelCount << endl;
	cout << "Progress = " << KernelP[arraySize - 1] - startLow << " at " << (KernelP[arraySize - 1] - startLow) / time_spent << " p/sec" << endl;

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_e);
	hipFree(dev_f);
	hipFree(dev_g);
	hipFree(dev_h);
	return cudaStatus;

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *NOut, unsigned long long *KernelP, unsigned int size, const int blocks, const int threads, unsigned long long *hashKeys, int *hashValues, int hashTableElements)
{

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_b, KernelP, size * sizeof(unsigned long long), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy input failed!");
    }

	cudaStatus = hipMemset(dev_g, 0, size * hashTableElements * 4 * sizeof(unsigned long long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy input failed!");
	}

	//cudaStatus = hipMemset(dev_h, 0, size * hashTableElements * 4 * sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy input failed!");
	//}


	//hipEvent_t start, stop;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);
    // Launch a kernel on the GPU with one thread for each element.
	//hipEventRecord(start);
	hipStream_t stream0;
	//hipStream_t stream1;
	hipStreamCreate(&stream0);
	//hipStreamCreate(&stream1);
    addKernel1<<<blocks,threads,0,stream0>>>(dev_a, dev_b, dev_c, dev_e, dev_f, dev_g, dev_h);
	//addKernel1<<<blocks,threads,0,stream1>>>(dev_a, dev_b, dev_c, dev_e, dev_f, dev_g, dev_h);
	//hipEventRecord(stop);

	//hipEventSynchronize(stop);
	//float milliseconds = 0;
	//hipEventElapsedTime(&milliseconds, start, stop);
	//printf("Time taken: %f ms \n", milliseconds);

    // Check for any errors launching the kernel
    //cudaStatus = hipGetLastError();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    //}
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(NOut, dev_a, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy output failed!");
    }
    
    return cudaStatus;
}
