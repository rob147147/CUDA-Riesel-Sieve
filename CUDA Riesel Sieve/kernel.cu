#include "hip/hip_runtime.h"
//Complete - Read in an ABCD file, find Q, split into subsequences
//Complete - CUDA code with correct outputs

//TODO - Scale hash table to the size of GPU RAM


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <list>

using namespace std;

hipError_t addWithCuda(int *NOut, unsigned long long *KernelP, unsigned int size, const int blocks, const int threads, int *hashKeys, int hashElements, int hashDensity, unsigned int *bits);
int *dev_a = 0; //NOut
unsigned long long *dev_b = 0; //KernelP
//int *dev_c = 0; //kns
__constant__ int dev_c[512];
int *dev_e; //Base
int *dev_f; //counterIn
int *dev_g = 0; //HashTable Keys
int *dev_h = 0; //HashTableElements
int *dev_i = 0; //HashTableDensity
unsigned int *dev_j = 0;

hipError_t cudaStatus;


__device__  __forceinline__ void xbinGCDnew(unsigned long long a, unsigned long long beta, unsigned long long &u, unsigned long long &v)
{
	unsigned long long alpha;
	u = 1; v = 0;
	alpha = a;
	// Note that alpha is
	// even and beta is odd.
	// The invariant maintained from here on is: 2a = u*2*alpha - v*beta.

	while (a > 0) {
		a = a >> 1;
		if ((u & 1) == 0) { // Delete a common
			u = u >> 1; v = v >> 1; // factor of 2 in
		} // u and v.
		else {
			/* We want to set u = (u + beta) >> 1, but
			that can overflow, so we use Dietz's method. */
			u = ((u ^ beta) >> 1) + (u & beta);
			v = (v >> 1) + alpha;
		}
	}
}


__device__ __forceinline__ void mulul64new(unsigned long long u, unsigned long long v, unsigned long long &wlo, unsigned long long &whi)
{
	unsigned long long u0, u1, v0, v1, k, t;
	unsigned long long w0, w1, w2;

	u1 = u >> 32; u0 = u & 0xFFFFFFFF;
	v1 = v >> 32; v0 = v & 0xFFFFFFFF;

	t = u0*v0;
	w0 = t & 0xFFFFFFFF;
	k = t >> 32;

	t = u1*v0 + k;
	w1 = t & 0xFFFFFFFF;
	w2 = t >> 32;

	t = u0*v1 + w1;
	k = t >> 32;

	wlo = (t << 32) + w0;
	whi = u1*v1 + w2 + k;

}

__device__ __forceinline__ unsigned long long modul64(unsigned long long x, unsigned long long y, unsigned long long z) {
	/* Divides (x || y) by z, for 64-bit integers x, y,
	and z, giving the remainder (modulus) as the result.
	Must have x < z (to get a 64-bit result). This is
	checked for. */
	long long t;
	if (x >= z) {
		printf("Bad call to modul64, must have x < z.");
	}
	for (int i = 1; i <= 64; i++) { // Do 64 times.
		t = (long long)x >> 63; // All 1's if x(63) = 1.
		x = (x << 1) | (y >> 63); // Shift x || y left      <- Bitwise OR?
		y = y << 1; // one bit.
		if ((x | t) >= z) {
			x = x - z;
			y = y + 1;
		}
	}
	return x; // Quotient is y.
}

__device__ __forceinline__ unsigned long long montmul(unsigned long long abar, unsigned long long bbar, unsigned long long m, unsigned long long mprime) {
	unsigned long long thi, tlo, tm;
	//unsigned long long uhi, ulo;
	unsigned int ov;

	//mulul64(abar, bbar, &thi, &tlo); // t = abar*bbar.
	thi = __umul64hi(abar, bbar);
	tlo = abar*bbar;
	/* Now compute u = (t + ((t*mprime) & mask)*m) >> 64.
	The mask is fixed at 2**64-1. Because it is a 64-bit
	quantity, it suffices to compute the low-order 64
	bits of t*mprime, which means we can ignore thi. */
	tm = tlo*mprime;
	//mulul64(tm, m, &tmmhi, &tmmlo); // tmm = tm*m.
	//tmmhi = __umul64hi(tm, m);
	//tmmlo = tm*m;


	//PTX Version 2 - Clobbers less registers - very similar speed.
	asm(//"{.reg .u64 t1;\n\t"              // temp 64-bit reg t1 = tmmlo
		"mad.lo.cc.u64 %0, %3, %4, %0;\n\t" //MAD: "tlo = (tm*m) + tlo" and set the carry out 
		//"add.cc.u64 %0, %0, t1;\n\t" //Add tlo = tlo + tmmlo and set carry out. 
		"madc.hi.cc.u64 %1, %3, %4, %1;\n\t" //MAD: "thi = hi(tm*m) + thi" use the previous carry and set the carry out
		//"addc.cc.u64 %1, %1, %3;\n\t" //Add thi and tmmhi, use the previous carry and set carry out.
		"addc.u32 %2, 0, 0;" //This sets ov to 1 if the previous addition overflowed.
		//"}"
		: "=+l"(tlo), "=+l"(thi) "=r"(ov) : "l"(tm), "l"(m)
		);

	if (ov > 0 || thi >= m) // If u >= m,
		thi = thi - m; // subtract m from u.
	return thi;
}

__device__ __forceinline__ long long binExtEuclid(long long a, long long b) {
	long long u = b;
	long long v = a;
	long long r = 0;
	long long s = 1;
	long long x = a;
	while (v>0) {
		if ((u & 1) == 0) {
			u = u >> 1;
			if ((r & 1) == 0) {
				r = r >> 1;
			}
			else {
				r = (r + b) >> 1;
			}
		}
		else {
			if ((v & 1) == 0) {
				v = v >> 1;
				if ((s & 1) == 0) {
					s = s >> 1;
				}
				else {
					s = (s + b) >> 1;
				}
			}
			else {
				x = u - v;
				if (x>0) {
					u = x;
					r = r - s;
					if (r<0) {
						r = r + b;
					}
				}
				else {
					v = x * -1;
					s = s - r;
					if (s<0) {
						s = s + b;
					}
				}
			}
		}
	}
	if (r >= b) {
		r = r - b;
	}
	if (r<0) {
		r = r + b;
	}
	return(r);
}



__global__ void addKernel1(int *NOut, unsigned long long *KernelP, /*int *ks,*/ int *Base, int *counterIn, int *hashKeys, int *hashElements, int *hashDensity, unsigned int *bits)
{
	clock_t beginfull = clock();
	clock_t begin = clock();

	//This deals with the hashTables
	const int m = *hashElements;
	const int mem = m * (*hashDensity); //This is hashTableElements * 4 to reduce collisions. Must be a power of 2
	const int ints = mem / 32;

	int shift = 0;
	int tempM = m;
	//m=2^shift, calculate shift
	while (tempM > 1) {
		tempM = tempM >> 1;
		shift++;
	}


	/*unsigned int bitArray[mem/32]; //Bit array for hash table

	for (int ii = 0; ii < 64; ii++) {
		bitArray[ii] = 0;
	}*/

	int S = (blockIdx.x * blockDim.x) + threadIdx.x; //This is this block ID*threads in a block + threadID
	int Sm = S*mem;

	bool printer = false;
	if (S == 0) {
		printer = true;
	}

	//extern __shared__ int bitArray[];

	//for (int ii = 0; ii < 64; ii++) {
	//	bitArray[threadIdx.x * 64 + ii] = 0;
	//}

	unsigned long long b = KernelP[S];
	int Q = dev_c[2];
	int NMin = dev_c[0]/Q;
	int NMax = (dev_c[1]/Q)+1;

	unsigned long long bprime = 0;
	unsigned long long rInv = 0;
	
	int montmuls = 0;
	int modul = 0;
	int mulul = 0;


	clock_t end = clock();
	int time_spent = (end - begin);
	if (printer) {
		printf("KernelBase = %d\n", *Base);
		printf("HashTableElements = %d. %d at 1/%d density.\n", mem,m,*hashDensity);
		printf("Each Thread should use %d ints in its bit array.\n", ints);
		printf("Q = %d, NMin = %d, NMax = %d\n", Q,NMin, NMax);
		printf("Cycles to complete variable setup was %d\n", time_spent);
	}

	begin = clock();

	xbinGCDnew(9223372036854775808, b, rInv, bprime);

	end = clock();
	time_spent = (end - begin);
	if (printer) {
		//Check GCD has worked
		printf("2*inp*rInv - b*bprime = %llu\n", (2 * 9223372036854775808 * rInv - (b*bprime)));
		printf("Cycles to do xbinGCD was %d\n", time_spent);
	}


	unsigned long long KernelBase = modul64(*Base, 0, b);
	modul++;
	unsigned long long newKB = modul64(1, 0, b);
	modul++;
	//We now deal with b^Q for subsequences. 
	for (int qq = 0; qq < Q; qq++) {
		newKB = montmul(KernelBase, newKB, b, bprime);
		montmuls++;
	}

	unsigned long long plo = 0;
	unsigned long long phi = 0;

	begin = clock();

	mulul64new(newKB, rInv, plo, phi);
	mulul++;

	end = clock();
	time_spent = (end - begin);

	if (printer) {
		printf("Cycles to do mulul64 was %d\n", time_spent);
	}

	newKB = modul64(phi, plo, b);
	modul++;
	
	unsigned long long newKB2 = newKB;

	begin = clock();

	newKB = binExtEuclid(newKB, b);

	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles to do binExtEuclid was %d\n", time_spent);
	}

	begin = clock();
	//Try to populate our new hash table array ------------------------------------------------------------
	int lookups = 0;
	int hash = 0;
	unsigned long long js = 1;

	//Convert js to montgomery space
	js = modul64(js, 0, b);
	modul++;
	//Convert newKB back into Montgomery space
	newKB = modul64(newKB, 0, b);
	modul++;

	if (printer) {
		end = clock();
		time_spent = (end - begin);
		printf("Cycles to do modul64 was %d\n", time_spent/2);
	}


	int maxProbeIn = 0;
	float avgProbeIn = 0;
	//int index = 0;
	for (int j = 0; j<m; j++) {

		clock_t beginindex;
		if (printer & j == m >> 1) {
			beginindex = clock();
		}

		hash = js & mem - 1;

		if (printer & j == m >> 1) {
			clock_t endindex = clock();
			time_spent = (endindex - beginindex);
			printf("Cycles to calculate hash and index was %d\n", time_spent);
		}

		//Basic linear probing
		for (int probe = 0; probe < m; probe++) {
			avgProbeIn++;
			if (probe > maxProbeIn) {
				maxProbeIn = probe;
			}
			lookups++;

			clock_t beginhash;
			if (printer & j == m >> 1) {
				beginhash = clock();
			}

			//if (hashKeys[(Sm + hash)] == 0) {
			if ((bits[S*ints + (hash / 32)] & (1 << (hash & 31))) == 0) {
				bits[S*ints + (hash / 32)] += 1 << (hash & 31);

				//Don't store it - we'll try and re-calculate it
				hashKeys[(Sm + hash)] = js; //This costs around 3750 cycles

				if (printer & j == m >> 1) {
					clock_t endhash = clock();
					time_spent = (endhash - beginhash);
					printf("Cycles to add key and value to hash table was %d\n", time_spent);
				}

				break;
			}

			hash = (hash + ((probe + 1)*(probe + 1))) & (mem - 1);

		}

		clock_t beginmul;
		if (printer & j == m >> 1) {
			beginmul = clock();
		}

		js = montmul(js, newKB, b, bprime);
		montmuls++;

		if (printer & j == m >> 1) {
			clock_t endmul = clock();
			time_spent = (endmul - beginmul);
			printf("Cycles to perform a montmul was %d\n", time_spent);
		}

	}
	if (printer) {
		printf("Number of baby steps: %d\n", m);
		printf("Number of lookups while inserting into the hash table was %d\n", lookups);
		printf("Max probe length for insert was %d\n", maxProbeIn);
		printf("Avg probe length for insert was %f\n", avgProbeIn / m);
	}


	//Finished calculating the hash table --------------------------------------------------------------------

	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles calculating new hash table was %d\n", time_spent);
		printf("Average was %d\n", time_spent / m);
	}

	begin = clock();
	//Compute KernelBase^-m (mod b)
	unsigned long long c1 = newKB2;
	c1 = modul64(newKB2, 0, b);
	modul++;

	//This should be KernelBase^-1 (mod b)
	//Now repeatedly square it as m is a power of two

	for (int t = 0; t<shift; t++) {
		c1 = montmul(c1, c1, b, bprime);
		montmuls++;
	}

	long long output = -5;

	int tMin = NMin >> shift;

	if (printer) {
		printf("tMin = %d\n", tMin);
	}

	lookups = 0;
	int countmuls = tMin;
	int giant = 0;
	int collisions = 0;

	int maxProbe = 0;
	float avgProbe = 0;

	unsigned long long fixedBeta = 0;
	unsigned long long fixedsB = modul64(1, 0, b);
	modul++;
	unsigned long long beta = 0;

	bool skip = false;
	int thisk = 0;
	//The first 3 values of ks contain NMin, NMax and Q now, so start at 3.
	for (int k = 3; k < *counterIn; k++) {
		if (skip) {
			skip = false;
		}

		else if (dev_c[k] == -1) {
			//This isn't a k-value, it proves the k-value is next
			thisk = dev_c[k+1];
			fixedBeta = modul64(thisk, 0, b);
			modul++;
			skip = true;
		}
		else {
			int remainder = dev_c[k];

			//Using subsequence k should be updated to be k*b^remainder
			unsigned long long sB = fixedsB;
			for (int rem = 0; rem < remainder; rem++) {
				sB = montmul(sB, KernelBase, b, bprime);
				montmuls++;
			}

			beta = montmul(fixedBeta, sB, b, bprime);
			montmuls++;

			for (int t = 0; t < tMin; t++) {
				beta = montmul(beta, c1, b, bprime);
				montmuls++;
			}

			if (printer & k == 0) {
				printf("We're in that bit that crashes!\n");
			}

			for (int t = tMin; t < (NMax/m)+1; t++) {
				giant++;

				//Check if beta is in js
				hash = beta & mem - 1;
				//index = hash*N + S;

				//Its possible beta is here, use linear probing to check
				for (int probe = 0; probe < m; probe++) {
					avgProbe++;
					if (probe > maxProbe) {
						maxProbe = probe;
					}
					lookups++;
					//if (hashKeys[(Sm + hash)] == 0) {
					if ((bits[S*ints + (hash / 32)] & (1 << (hash & 31))) == 0) {
						//Beta is not here
						break;
					}
					collisions++;

					if (hashKeys[Sm + hash] == (int)beta) {

						lookups++;
						//printf("Match in Thread %d, Block %d. t=%d, hash=%d, probe=%d beta=%llu. Output will be %llu | %d*%d^%d-1\n", i, block, t, hash, probe, beta, b, ks[k], outputBase, output);

						//We've found beta
						//We've had a match
						//Find the j value
						unsigned long long jsnew = 1;
						jsnew = modul64(jsnew, 0, b);
						modul++;
						for (int jval = 0; jval < m; jval++) {
							if (jsnew == beta) {
								output = t*m + jval;
								break;
							}
							jsnew = montmul(jsnew, newKB, b, bprime);
							montmuls++;
						}
						//printf("Match in Thread %d, Block %d. t=%d, hash=%d, probe=%d beta=%llu. Output will be %llu | %d*%d^%d-1\n", i, block, t, hash, probe, beta, b, ks[k], outputBase, output);
						break;
					}

					hash = (hash + ((probe + 1)*(probe + 1))) & (mem - 1);
				}

				beta = montmul(beta, c1, b, bprime);
				countmuls++;
				montmuls++;
			}

			if (output < NMin) {
				output = -3;
			}
			else if (output > NMax) {
				output = -4;
			}
			else {
				printf("Output will be %llu | %d*%d^%d-1\n", b, thisk, *Base, (output*Q)+ dev_c[k]);
				output = -5;
			}
		}

	}
	if (printer) {
		printf("Number of giant steps: %d\n", giant);
		printf("Number of collisions: %d\n", collisions);
		printf("Number of lookups against hash table was %d\n", lookups);
		printf("Max probe length was %d\n", maxProbe);
		printf("Average probe length was %f\n", avgProbe / giant);
	}

	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles to complete BSGS step was %d\n", time_spent);
		printf("Average (BSGS Cycles/muls) was %d\n", (time_spent / countmuls));
		printf("Average (BSGS Cycles/lookups) was %d\n", (time_spent / lookups));
	}

	begin = clock();

	NOut[S] = output; //This should contain the k-value in the top 32 bits and the n-value in the low 32 bits

	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles to write output to NOut was %d\n", time_spent);
		printf("Total number of montgomery multiplies was %d\n", montmuls);
		printf("Total number of calls to modul64 was %d\n", modul);
		printf("Total number of calls to mulul64 was %d\n", mulul);
	}

	time_spent = (end - beginfull);
	if (printer) {
		printf("Cycles to execute one full thread was %d\n", time_spent);
	}
}


//Update this at some point to use getopt
int main(int argc, char* argv[])
{
	const int kb = 1024;
	const int mb = kb * kb;

	wcout << "CUDA version:   v" << CUDART_VERSION << endl;

	int devCount;
	hipGetDeviceCount(&devCount);
	wcout << "CUDA Devices: " << devCount << endl;

	hipDeviceProp_t props;

	for (int i = 0; i < devCount; ++i)
	{
		hipGetDeviceProperties(&props, i);
		wcout << props.name << ":" << endl;
		wcout << "  CC: " << props.major << "." << props.minor << endl;
		wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
		wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
		wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
		wcout << "  Block registers: " << props.regsPerBlock << endl << endl;

		wcout << "  Warp size:         " << props.warpSize << endl;
		wcout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
		wcout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << " ]" << endl;
		wcout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1] << ", " << props.maxGridSize[2] << " ]" << endl;
		wcout << "  L2 Cache Size: " << props.l2CacheSize / kb << "kb" << endl;
		wcout << endl;
	}

	unsigned long long targetHashSize = 1;
	while (targetHashSize < props.totalGlobalMem / 2) {
		targetHashSize = targetHashSize << 1;
	}
	wcout << "Should target a " << targetHashSize / mb << "mb hash table" << endl << endl;

	//Read in an ABCD file and parse ----------------------------------------------------------------------------
	string line;
	int total = 0;
	//string abcdFile = "C:\\Users\\Rob\\Documents\\Visual Studio 2015\\Projects\\CPU Sieve\\sr_108.abcd";
	//string abcdFile = "C:\\Users\\Rob\\Desktop\\TestSieve\\sr_745.abcd";
	string abcdFile = "sr_745.abcd";

	//First pass through the ABCD file to find the number of k's and max number of n's
	int count1 = 0; //Number of k's
	int count3 = 0; //Total number of lines
	ifstream myfile(abcdFile);
	if (myfile.is_open())
	{
		while (getline(myfile, line))
		{
			count3++;

			string::size_type n = line.find(" ");
			string token = line.substr(0, n);
			//cout << token << endl;

			//If tokens[0] == "ABCD" then this defines a new k, otherwise it is a number
			if (token.compare("ABCD") == 0) {
				count1++;
				//cout << "We're here!" << endl;
			}
		}
		myfile.close();
	}

	else cout << "Unable to open file first time" << endl;

	//Second pass through the ABCD file to write the values into the matrix
	//Store the k and n values in this array
	//boost::numeric::ublas::matrix<int> kns(count1, max);
	//std::list<int> kns;
	count3 = count3 + (2 * count1);
	int *kns = (int *)malloc(count3*sizeof(int));
	int *ks = (int *)malloc(count1*sizeof(int));

	int minN = INT_MAX;
	int maxN = 0;

	//Reset the counts
	count1 = 0;
	count3 = 0;
	int base = 0;
	ifstream myfile2(abcdFile);
	if (myfile2.is_open())
	{
		while (getline(myfile2, line))
		{
			//Tokenise the string - if the first element of the string is "ABCD" then this is a new k-value
			string::size_type n = line.find(" ");
			string token = line.substr(0, n);
			//If tokens[0] == "ABCD" then this defines a new k, otherwise it is a number
			if (token.compare("ABCD") == 0) {
				cout << "We've found ABCD. Get the k-value" << endl;
				//Insert a 0 into kns before we insert the k-value
				kns[count3] = 0;
				count3++;
				//Get the k value
				token = line.substr(n + 1);
				//cout << token << endl;
				n = token.find("*");
				string tok = token.substr(0, n);
				//cout << tok << endl;

				int kval = stoi(tok);
				kns[count3] = kval;
				count3++;
				ks[count1] = kval;
				count1++;
				//Get the base
				if (base == 0) {
					token = token.substr(n + 1);
					n = token.find("^");
					string b = token.substr(0, n);
					//cout << b << endl;
					base = stoi(b);
					cout << "The base is " << base << endl;
				}
				//Get the starting n-value - remove the square brakets
				n = token.find("[");
				token = token.substr(n + 1);
				n = token.find("]");
				token = token.substr(0, n);
				//cout << token << endl;
				total = stoi(token);
				if (total < minN) {
					minN = total;
				}
				kns[count3] = total;
				count3++;
				cout << "This is a new k-value with value " << kval << " and initial n-value " << total << endl;
			}
			else {
				//This is a number, n-value offset
				//cout << token << endl;
				int offset = stoi(token);
				total = total + offset;
				if (total > maxN) {
					maxN = total;
				}
				kns[count3] = total;
				count3++;
				//cout << count3 << endl;
			}

		}

		myfile2.close();
	}

	else cout << "Unable to open file second time" << endl;

	//End of reading ABCD file ----------------------------------------------------------------------------------
	cout << "End of reading ABCD file" << endl;
	//-----------------------------------------------------------------------------------------------------------

	//Find the optimal Q value, and use subsequences with base b^Q ----------------------------------------------
	cout << "Find the optimal(ish) Q value for subsequences in base b^Q" << endl;
	int range = maxN - minN;
	int minWork = count1*range;
	int minSubs = count1;
	int minRange = range;
	int minQ = 0;
	cout << count1 << " k-values over a range of " << range << " n-values" << endl;
	cout << "Q=1 work: " << minWork << endl;
	//Iterate through Q values - from 2 to 24(?)
	for (int Q = 2; Q < 24; Q++) {
		int subsequences = 0;
		//Count the number of subsequences
		bool *subseq = (bool *)malloc(count1*Q*sizeof(bool));
		memset(subseq, false, count1*Q*sizeof(bool));
		int whichk = -1;
		for (int qq = 0; qq < count3; qq++) {
			if (kns[qq] == 0) {
				whichk++;
				//Skip the k-value itself which is next in the array
				qq++;
			}
			else {
				//This is an n-value
				int mod = kns[qq] % Q;
				subseq[whichk*Q + mod] = true;
			}
		}
		//Count the number of true's in subseq - this is our number of subsequences if we use Q
		for (int qqq = 0; qqq < count1*Q; qqq++) {
			if (subseq[qqq] == true) {
				subsequences++;
			}
		}

		range = (maxN / Q) - (minN / Q) + 1;
		if (subsequences*range < minWork) {
			minQ = Q;
			minWork = subsequences*range;
			minSubs = subsequences;
			minRange = range;
		}
		cout << "Q=" << Q << ". Work= " << subsequences*range << ". Range = " << range << ". Subsequences= " << subsequences << endl;
		free(subseq);
	}

	//Successfully found Q --------------------------------------------------------------------------------------
	cout << "Min Work has Q=" << minQ << ". Work= " << minSubs*minRange << ". Range = " << minRange << ". Subsequences= " << minSubs << endl;

	//Recalculate the boolean array for our chosen Q
	bool *subseq = (bool *)malloc(count1*minQ*sizeof(bool));
	memset(subseq, false, count1*minQ*sizeof(bool));
	int whichk = -1;
	for (int qq = 0; qq < count3; qq++) {
		if (kns[qq] == 0) {
			whichk++;
			//Skip the k-value itself which is next in the array
			qq++;
		}
		else {
			//This is an n-value
			int mod = kns[qq] % minQ;
			subseq[whichk*minQ + mod] = true;
		}
	}

	//Edit the list of k-values to now contain the k and its associated modulo values based on Q
	//Use ks2 for now to store nMin and nMax too. 
	int *ks2 = (int *)malloc((count1*2 + minSubs + 3)*sizeof(int));
	ks2[0] = minN;
	ks2[1] = maxN;
	ks2[2] = minQ;
	int ks2counter = 3;
	for (int kvalues = 0; kvalues < count1; kvalues++) {
		//Seperate k values with a -1 marker
		ks2[ks2counter] = -1;
		ks2counter++;

		ks2[ks2counter] = ks[kvalues];
		ks2counter++;
		for (int Q = 0; Q < minQ; Q++) {
			if (subseq[kvalues*minQ + Q] == true) {
				ks2[ks2counter] = Q;
				ks2counter++;
			}
		}
	}

	free(subseq);


	//Generate Primes -------------------------------------------------------------------------------------------

	int blockScale = 32; //Default scaling
	int threadScale = 2; //Default scaling 
	//Use the input arguments to change blocks and threads. 
	if (argc == 2) {
		//Assume only a threadScale - we must check this is a power of 2 at some point
		threadScale = atoi(argv[1]);
	}
	if (argc == 3) {
		//Assume threadScale and then blockScale - - we must check these are both a power of 2 at some point
		threadScale = atoi(argv[1]);
		blockScale = atoi(argv[2]);
	}

	const int blocks = 32 * blockScale;
	const int threads = 32 * threadScale; //These must multiply to around 65536. Larger and CUDA times out
	const int arraySize = blocks*threads;
	const int testArraySize = arraySize * 24;
	const int hashScaling = 4;

	//Use targetHashSize to set up the hash table - int = 32 bits = 4 bytes, so divide by 4
	//Each thread requires the a hash table, so also divide by arraySize
	long long longhashTableSize = (((targetHashSize / 4) / arraySize)/hashScaling);
	int hashTableSize = longhashTableSize;
	cout << "Each thread should have " << hashTableSize*hashScaling << " buckets, to store " << hashTableSize << " elements. (Density 1/" << hashScaling << ")" << endl;

	unsigned long long *KernelP = (unsigned long long *)malloc(arraySize*sizeof(unsigned long long));
	int *NOut = (int *)malloc(arraySize*sizeof(int));
	int *hashKeys = (int *)malloc(arraySize * hashTableSize * hashScaling * sizeof(int));
	unsigned int *bits = (unsigned int *)malloc(((arraySize * hashTableSize * hashScaling)/32) * sizeof(int));
	memset(hashKeys, 0, arraySize * hashTableSize * hashScaling * sizeof(int));
	memset(bits, 0, ((arraySize * hashTableSize * hashScaling)/32) * sizeof(int));


	//Low should be greater than the primes we use below. 
	//unsigned long long low = 6000000000;
	//unsigned long long high = 6004000000;

	//unsigned long long low = 1000067500000;
	//unsigned long long high = 1000070000000;

	//unsigned long long low = 1000099000000;
	//unsigned long long high = 1000100000000;

	//unsigned long long low = 102254819500000L;
	unsigned long long low = 102297149770000L;
	unsigned long long high = 102297160000000L;

	//unsigned long long low = 600000;
	//unsigned long long high = 10000000;


	unsigned long long startLow = low; //Don't touch this. Used for timing purposes

	//Use the idea of a segmented sieve. Generate a list of small primes first
	//Could use the first 1024 primes as a starter. 8161 is the 1024th prime

	clock_t begin = clock();
	int smallPrimes = 8162;
	int primeCount = 1024;
	int s = 0;
	bool *primes = (bool *)malloc(smallPrimes*sizeof(bool));
	unsigned int *smallP = (unsigned int *)malloc(primeCount*sizeof(unsigned int));
	memset(primes, true, smallPrimes*sizeof(bool));

	int sq = smallPrimes*smallPrimes;

	for (int p = 2; p*p < sq; p++) {
		if (primes[p] == true) {
			smallP[s] = p;
			//cout << smallP[s] << endl;
			s++;
			for (int i = p * 2; i < smallPrimes; i += p) {
				primes[i] = false;
			}
		}
	}
	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	cout << "Time generating small primes " << time_spent << "s" << endl;

	//Print the small primes as a check
	for (int p = 0; p < primeCount; p++) {
		//cout << smallP[p] << endl;
	}

	//Find the minimum number in [low...high] that is a multiple of primes[i]

	bool *mark = (bool *)malloc(testArraySize*sizeof(bool));



	//Try setting up the GPU just once

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (one output, seven input). 
	//Give all vectors same size for now, we can change this afterwards

	cudaStatus = hipMalloc((void**)&dev_a, arraySize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, arraySize * sizeof(unsigned long long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//cudaStatus = hipMalloc((void**)&dev_c, (count1 * 2 + minSubs + 3) * sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//	goto Error;
	//}

	cudaStatus = hipMalloc((void**)&dev_e, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_f, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_g, arraySize * hashTableSize * hashScaling * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_h, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_i, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_j, ((arraySize * hashTableSize * hashScaling) / 32) * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//Copy the data to the correct GPU buffers

	//Lets try storing the k values and remainders in constant memory instead
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(dev_c), ks2, (count1 * 2 + minSubs + 3) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy to constant memory failed!");
		cout << (count1 * 2 + minSubs + 3) * sizeof(int) << "bytes" << endl;
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_e, &base, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_f, &ks2counter, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_g, hashKeys, arraySize * hashTableSize * hashScaling * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy input failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_h, &hashTableSize, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_i, &hashScaling, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_j, bits, ((arraySize * hashTableSize * hashScaling) / 32) * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	int kernelCount = 0;
	clock_t loopTime = clock();
	//From here we need to loop to keep the GPU busy. 
	while (low < high) {
		kernelCount++;
		cout << "Executing kernel number " << kernelCount << endl;
		cout << "Low is now set to " << low << endl;

		begin = clock();
		memset(mark, true, testArraySize*sizeof(bool));

		for (int i = 0; i < primeCount; i++) {
			unsigned int smallPrime = smallP[i];
			for (int j = 0; j < testArraySize; j++) {
				//if (mark[j] == true && (((low + j) % smallP[i]) == 0)) {
				if (((low + j) % smallPrime) == 0) {
					//So if low + offset can be divided by i we've found the first value divisible by i. Now mark off all i multiples
					for (int k = j; k < testArraySize; k += smallPrime) {
						mark[k] = false;
					}
					break;
				}
			}
		}
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		cout << "Time marking the prime array " << time_spent << "s" << endl;


		// Numbers which are not marked as false are prime
		begin = clock();
		int countPrimes = 0;
		for (unsigned long long i = low; i < low + (testArraySize); i++) {
			if (mark[i - low] == true) {
				KernelP[countPrimes] = i;
				countPrimes++;
				if (countPrimes == arraySize) {
					cout << "We got as far as " << i << " out of " << low + (testArraySize) << endl;
					break;
				}
				//cout << i << endl;
			}
		}
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		cout << "Time generating kernel primes " << time_spent << "s" << endl;

		unsigned long long minPrime = KernelP[0];
		unsigned long long maxPrime = KernelP[arraySize - 1];
		unsigned long long progress = maxPrime - minPrime;

		cout << "Min Prime = " << minPrime << ". Max Prime = " << maxPrime << ". Progress = " << progress << endl;
		cout << "Array Size = " << arraySize << endl;

		//End of Generating Primes ----------------------------------------------------------------------------------

		begin = clock();
		cout << "Try to launch the CUDA kernel" << endl;
		// Add vectors in parallel.
		//This uses the full ABCD file, but runs very slowly when file is big
		//hipError_t cudaStatus = addWithCuda(NOut, KernelP, kns, &base, &count3, arraySize, count3, blocks, threads);
		//This is datless - remember to change to addkernel1
		hipError_t cudaStatus = addWithCuda(NOut, KernelP, arraySize, blocks, threads, hashKeys, hashTableSize, hashScaling, bits);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		cout << "Time to execute kernel (outside function) " << time_spent << "s" << endl;

		printf("%d,%d,%d,%d,%d,%d,%d,%d,%d,%d\n\n",
			NOut[0], NOut[1], NOut[2], NOut[3], NOut[4], NOut[5], NOut[6], NOut[7], NOut[8], NOut[9]);

		//Set low equal to high and continue in the loop
		low = maxPrime;
	}

	clock_t loopEnd = clock();
	time_spent = (double)(loopEnd - loopTime) / CLOCKS_PER_SEC;
	cout << "Time taken " << time_spent << "s" << endl;
	cout << "Time per kernel " << time_spent / kernelCount << endl;
	cout << "Progress = " << KernelP[arraySize - 1] - startLow << " at " << (KernelP[arraySize - 1] - startLow) / time_spent << " p/sec" << endl << endl;

	//Reprint the CUDA info
	wcout << "CUDA version:   v" << CUDART_VERSION << endl;

	hipGetDeviceCount(&devCount);
	wcout << "CUDA Devices: " << devCount << endl;

	for (int i = 0; i < devCount; ++i)
	{
		hipGetDeviceProperties(&props, i);
		wcout << props.name << ":" << endl;
		wcout << "  CC: " << props.major << "." << props.minor << endl;
		wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
		wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
		wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
		wcout << "  Block registers: " << props.regsPerBlock << endl;
		wcout << "  L2 Cache Size: " << props.l2CacheSize / kb << "kb" << endl;
		wcout << endl;
	}

	cout << "Each thread used " << hashTableSize*hashScaling << " buckets, to store " << hashTableSize << " elements. (Density 1/" << hashScaling << ")" << endl;
	cout << "Hash table size was " << (hashTableSize*hashScaling * 4 * arraySize) / mb << "mb of GPU RAM" << endl;
	cout << "Blocksize = " << blocks << ". Threads per block = " << threads << "." << endl;

Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_e);
	hipFree(dev_f);
	hipFree(dev_g);

	hipFree(dev_j);
	return cudaStatus;

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *NOut, unsigned long long *KernelP, unsigned int size, const int blocks, const int threads, int *hashKeys, int hashElements, int hashDensity, unsigned int *bits)
{

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_b, KernelP, size * sizeof(unsigned long long), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy input failed!");
	}

	cudaStatus = hipMemset(dev_g, 0, size * hashElements * hashDensity * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy input failed!");
	}

	cudaStatus = hipMemset(dev_j, 0, ((size * hashElements * hashDensity) / 32) * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy input failed!");
	}


	//hipEvent_t start, stop;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);
	// Launch a kernel on the GPU with one thread for each element.
	//hipEventRecord(start);
	hipStream_t stream0;
	//hipStream_t stream1;
	hipStreamCreate(&stream0);
	//hipStreamCreate(&stream1);
	addKernel1 <<<blocks, threads, 0, stream0 >>>(dev_a, dev_b, /*dev_c, */dev_e, dev_f, dev_g, dev_h, dev_i, dev_j);
	
	//This uses too much shared memory and kills occupancy. Really we want to use no more than 16 ints per thread (for 64 threads per block)!
	//addKernel1 << <blocks, threads, ((threads*hashElements*hashDensity) / 32)*sizeof(int), stream0 >> >(dev_a, dev_b, dev_c, dev_e, dev_f, dev_g, dev_h, dev_i);

	//addKernel1<<<blocks,threads,0,stream1>>>(dev_a, dev_b, dev_c, dev_e, dev_f, dev_g, dev_h);
	//hipEventRecord(stop);

	//hipEventSynchronize(stop);
	//float milliseconds = 0;
	//hipEventElapsedTime(&milliseconds, start, stop);
	//printf("Time taken: %f ms \n", milliseconds);

	// Check for any errors launching the kernel
	//cudaStatus = hipGetLastError();
	//if (cudaStatus != hipSuccess) {
	//    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	//}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}

	// Copy output vector from GPU buffer to host memory.
	//cudaStatus = hipMemcpy(NOut, dev_a, size * sizeof(int), hipMemcpyDeviceToHost);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy output failed!");
	//}

	return cudaStatus;
}

