#include "hip/hip_runtime.h"
//Complete - Read in an ABCD file, find Q, split into subsequences
//Complete - CUDA code with correct outputs

//TODO - Scale hash table to the size of GPU RAM


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <getopt.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <list>

#include "FileInput.h"

using namespace std;

void generateGPUPrimes(unsigned long long *KernelP, unsigned long long low, unsigned int *smallP, int testArraySize, int primeCount, int arraySize, unsigned int *mark1);

#define PRINT true
//#undef PRINT

int *dev_a = 0; //NOut
unsigned long long *dev_b = 0; //KernelP
int *dev_c = 0; //kns
				//__constant__ int dev_c[512];
__constant__ int base[1]; //Base
int *dev_f; //counterIn
int *dev_g = 0; //HashTable Keys
int *dev_h = 0; //HashTableElements
int *dev_i = 0; //HashTableDensity
unsigned int *dev_j = 0;
int *dev_k = 0; //Q
__constant__ int tMin[1]; //tMin
__constant__ int tMax[1]; //tMax
int *dev_n = 0; //minSubs

hipError_t cudaStatus;


__device__ __forceinline__ int legendre(unsigned int a, unsigned long long p) {
	//Work out the legendre symbol for (a/p)
	//This code is taken straight from the source code of SR2Sieve
	unsigned int x, y;
	//Odd sign is positive(sign&1==1), even sign is negative(sign&1==0)
	unsigned int sign = 1;
	for (y = a; y % 2 == 0; y /= 2) {
		if (p % 8 == 3 || p % 8 == 5) {
			sign++;
		}
	}
	if (p % 4 == 3 && y % 4 == 3) {
		sign++;
	}

	unsigned long long xtemp = p % y;

	for (x = int(xtemp); x>0; x %= y) {
		for (; x % 2 == 0; x /= 2) {
			if (y % 8 == 3 || y % 8 == 5) {
				sign++;
			}
		}
		//Swap x and y
		//t = x, x = y, y = t;
		x = x ^ y;
		y = x ^ y;
		x = x ^ y;

		if (x % 4 == 3 && y % 4 == 3) {
			sign++;
		}
	}

	return sign & 1;
}


__device__  __forceinline__ void xbinGCDnew(unsigned long long beta, unsigned long long &u, unsigned long long &v)
{
	unsigned long long alpha = 9223372036854775808;
	//unsigned long long a = 9223372036854775808;
	//u = 1; v = 0;
	//alpha = a;
	// Note that alpha is
	// even and beta is odd.
	// The invariant maintained from here on is: 2a = u*2*alpha - v*beta.

	//while (a > 0) { //This is just a counter as a is never used. 
	//	a = a >> 1;
	#pragma unroll 1
	for (int i=0; i<64; i++) {
		if ((u & 1) == 0) { // Delete a common
			u = u >> 1; v = v >> 1; // factor of 2 in
		} // u and v.
		else {
			/* We want to set u = (u + beta) >> 1, but
			that can overflow, so we use Dietz's method. */
			u = ((u ^ beta) >> 1) + (u & beta);
			v = (v >> 1) + alpha; //v>>1 happens in both cases, this just also sets the highest bit to 1
		}
	}
}

int core(unsigned int k) {
	//Return the square free part of k
	//Basic method - if remainder after dividing by a prime is 0 twice, we can remove this from k
	//At the moment just remove a single 2^2, 3^2 and/or 5^2
	if (k % 2 == 0) {
		k = k / 2;
		if (k % 2 == 0) {
			k = k / 2;
		}
		else {
			k = k * 2;
		}
	}
	if (k % 3 == 0) {
		k = k / 3;
		if (k % 3 == 0) {
			k = k / 3;
		}
		else {
			k = k * 3;
		}
	}
	if (k % 5 == 0) {
		k = k / 5;
		if (k % 5 == 0) {
			k = k / 5;
		}
		else {
			k = k * 5;
		}
	}
	return k;
}


__device__ __forceinline__ unsigned long long modul64(unsigned long long x, unsigned long long y, unsigned long long z) {
	/* Divides (x || y) by z, for 64-bit integers x, y,
	and z, giving the remainder (modulus) as the result.
	Must have x < z (to get a 64-bit result). This is
	checked for. */

	//If we limit z to being less 2^63, then x will always have a 0 first bit (as x < z)
	//In which case t will always be 0, so is not needed. 
	//Even when we shift x left (double it) after we subtract z it will never have its first bit set. 


	//long long t;
	#ifdef PRINT
	if (x >= z) {
		printf("Bad call to modul64, must have x < z.");
	}
	#endif

#pragma unroll 1
	for (int i = 0; i < 64; i++) { // Do 64 times.
		//t = (long long)x >> 63; // All 1's if x(63) = 1?
		x = (x << 1) | (y >> 63); // Shift x || y left      <- Bitwise OR?
		y = y << 1; // one bit.
		//if ((x | t) >= z) {
		if (x >= z) {
			x = x - z;
			y = y + 1;
		}
	}
	return x; // Quotient is y.
}


__device__ __forceinline__ unsigned long long montmul(unsigned long long abar, unsigned long long bbar, unsigned int mlo, unsigned int mhi, unsigned int mprimelo, unsigned int mprimehi) {

	//Take the 64 bit inputs, but do all multiplies in 32 bit chunks
	unsigned int alo = (unsigned int)abar;
	unsigned int ahi = (unsigned int)(abar >> 32);
	unsigned int blo = (unsigned int)bbar;
	unsigned int bhi = (unsigned int)(bbar >> 32);

	unsigned int u0 = 0;
	unsigned int u1 = 0;
	unsigned int u2 = 0;
	unsigned int u3 = 0;

	unsigned int tm0 = 0;
	unsigned int tm1 = 0;

	//We need to calculate all 128 bits of t = abar * bbar. Add straight to u
	//PTX Version 1
	asm("mul.hi.u32 %1, %6, %8;\n\t" //Bits 32-64 (u1)
		"mul.lo.u32 %2, %7, %9;\n\t" //Bits 65-96 (u2)
		"mul.lo.u32 %0, %6, %8; \n\t" //Lowest 32 bits of u (u0)
		"mad.lo.cc.u32 %1, %6, %9, %1;\n\t" //Add the crossproduct to u1. Set the carry out
		"madc.hi.cc.u32 %2, %6, %9, %2;\n\t" //Add the crossproduct to u2. Set the carry out, and use the carry in
		"madc.hi.u32 %3, %7, %9, %3;\n\t" //Bits 97-128 plus any carry out (u3)
		"mad.lo.cc.u32 %1, %7, %8, %1;\n\t" //Add the other crossproduct to u1. Set the carry out
		"madc.hi.cc.u32 %2, %7, %8, %2;\n\t" //Add the other crossproduct to u2. Set the carry out, and use the carry in
		"addc.u32 %3, %3, 0;\n\t" //Add the potential carry into u3

	//Now we calculate t*mprime & mask. I.e. the low 64 bits
		"mul.hi.u32 %5, %0, %12;\n\t" //tm1
		"mul.lo.u32 %4, %0, %12;\n\t" //tm0 = lo part of u0*mprimelo
		"mad.lo.u32 %5, %0, %13, %5;\n\t" //Add the lo part of the crossproduct to tm1. u0*mprimehi
		"mad.lo.u32 %5, %1, %12, %5;\n\t" //Add the lo part of the other crossproduct to tm1 u1*mprimelo

	//Multiple tm*m to get the 128 bit product, and add it to u
		"mad.lo.cc.u32 %0, %4, %10, %0;\n\t" //Add the lo part of tm.lo*m*lo to u0. Set the carry out
		"madc.hi.cc.u32 %1, %4, %10, %1;\n\t" //Add the hi part of tm.lo*m*lo to u1. Set the carry out, and use the carry in
		"madc.lo.cc.u32 %2, %5, %11, %2;\n\t" //Add the lo part of tm.hi*m*hi to u2. Set the carry out, and use the carry in
		"madc.hi.cc.u32 %3, %5, %11, %3;\n\t" //Add the hi part of tm.hi*m*hi to u3. Use the carry in, and set carry out for overflow detection
		"addc.u32 %0, 0, 0;\n\t" //Put the overflow flag into u0
		"mad.lo.cc.u32 %1, %4, %11, %1;\n\t" //Add the lo part of tm.lo*m*hi to u1. Set the carry out
		"madc.hi.cc.u32 %2, %4, %11, %2;\n\t" //Add the hi part of tm.lo*m*hi to u2. Set the carry out, and use the carry in
		"addc.cc.u32 %3, %3, 0;\n\t" //Add the potential carry into u3
		"addc.u32 %0, %0, 0;\n\t" //Put the overflow flag into u0
		"mad.lo.cc.u32 %1, %5, %10, %1;\n\t" //Add the lo part of tm.hi*m*lo to u1. Set the carry out
		"madc.hi.cc.u32 %2, %5, %10, %2;\n\t" //Add the hi part of tm.hi*m*lo to u2. Set the carry out, and use the carry in
		"addc.cc.u32 %3, %3, 0\n\t;" //Add the potential carry into u3
		"addc.u32 %0, %0, 0;" //Put the overflow flag into u0

		: "=+r"(u0), "=+r"(u1), "=+r"(u2), "=+r"(u3), "=+r"(tm0), "=+r"(tm1) : "r"(alo), "r"(ahi), "r"(blo), "r"(bhi), "r"(mlo), "r"(mhi), "r"(mprimelo), "r"(mprimehi)
	);

	//We're only interested in u2 and u3
	unsigned long long u = u3;
	u = u << 32;
	u = u | u2;

	unsigned long long m = mhi;
	m = m << 32;
	m = m | mlo;

	if (u0 > 0 || u >= m) {
		u = u - m;
	}
	return u;

}

__device__ __forceinline__ long long binExtEuclid(long long a, long long b) {
	long long u = b;
	long long v = a;
	long long r = 0;
	long long s = 1;
	long long x = a;
	while (v>0) {
		if ((u & 1) == 0) {
			u = u >> 1;
			r = (r + ((r & 1)*b)) >> 1;
			//if ((r & 1) == 0) {
			//	r = r >> 1;
			//}
			//else {
			//	r = (r + b) >> 1;
			//}
		}
		else {
			if ((v & 1) == 0) {
				v = v >> 1;
				s = (s + ((s & 1)*b)) >> 1;
				//if ((s & 1) == 0) {
				//	s = s >> 1;
				//}
				//else {
				//	s = (s + b) >> 1;
				//}
			}
			else {
				x = u - v;
				if (x>0) {
					u = x;
					r = r - s;
					if (r<0) {
						r = r + b;
					}
				}
				else {
					v = x * -1;
					s = s - r;
					if (s<0) {
						s = s + b;
					}
				}
			}
		}
	}
	if (r >= b) {
		r = r - b;
	}
	if (r<0) {
		r = r + b;
	}
	return(r);
}



__global__ void addKernel1(int *NOut, unsigned long long *KernelP, int *knmatrix, int *rowOffset, int *hashKeys, int *hashElements, int *hashDensity, unsigned int *bits, int *Q, int *minSubs)
{
	clock_t beginfull = clock();
	clock_t begin = clock();

	int legtime = 0;
	clock_t beginLeg = clock();
	clock_t endLeg = clock();

	clock_t end = clock();
	int time_spent = 0;

	//This deals with the hashTables
	const int m = *hashElements;
	const int mem = m * (*hashDensity); //This is hashTableElements*density, to keep the correct thread using correct hash table 

	//m=2^shift, calculate shift
	int shift = 31 - __clz(m);

	const int S = (blockIdx.x * blockDim.x) + threadIdx.x; //This is this block ID*threads in a block + threadID
	const int Sm = S * mem;
	const int Sints = Sm >> 5;

	bool printer = false;
	if (S == 0) {
		printer = true;
	}

	const unsigned long long b = KernelP[S];
	const unsigned long long oneMS = modul64(1, 0, b);

	unsigned long long bprime = 0;
	unsigned long long rInv = 1;

	#ifdef PRINT
	int montmuls = 0;
	int montmuls1 = 0;
	int modul = 0;
	int bitLookups = 0;
	int bitUpdates = 0;
	int inserts = 0;

	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("KernelBase = %d\n", base[0]);
		printf("HashTableElements = %d. %d at 1/%d density.\n", mem, m, *hashDensity);
		//printf("Each Thread should use %d ints in its bit array.\n", ints);
		printf("Q = %d.\n", *Q);
		printf("Cycles to complete variable setup was %d\n", time_spent);
	}

	begin = clock();
	#endif

	xbinGCDnew(b, rInv, bprime);

	#ifdef PRINT
	end = clock();
	time_spent = (end - begin);
	if (printer) {
		//Check GCD has worked
		printf("2*inp*rInv - b*bprime = %llu\n", (2 * 9223372036854775808 * rInv - (b*bprime)));
		printf("Cycles to do xbinGCD was %d\n", time_spent);
	}
	#endif

	//beginModul = clock();
	unsigned long long KernelBase = modul64(base[0], 0, b);
	unsigned long long newKB = oneMS;

	//endModul = clock();
	//time_spent = (endModul - beginModul);
	//modultime += time_spent;
	//modul = modul + 2;

	unsigned int mlo = (unsigned int)b;
	unsigned int mhi = (unsigned int)(b >> 32);
	unsigned int mprimelo = (unsigned int)bprime;
	unsigned int mprimehi = (unsigned int)(bprime >> 32);

	//We now deal with b^Q for subsequences. 
	for (int qq = 0; qq < *Q; qq++) {
		//beginMont = clock();
		newKB = montmul(KernelBase, newKB, mlo, mhi, mprimelo, mprimehi);
		//newKB = montmul64(KernelBase, newKB, b, bprime);
		//endMont = clock();
		//time_spent = (endMont - beginMont);
		//montmultime += time_spent;
		//montmuls++;
	}

	//Save this now so we can use it later. Will save a call to modul
	unsigned long long c1 = newKB;

	unsigned long long plo = newKB * rInv;
	unsigned long long phi = __umul64hi(newKB, rInv);


	//beginModul = clock();
	newKB = modul64(phi, plo, b);
	//endModul = clock();
	//time_spent = (endModul - beginModul);
	//modultime += time_spent;
	//modul++;

	//unsigned long long newKB2 = newKB;

	#ifdef PRINT
	begin = clock();
	#endif

	newKB = binExtEuclid(newKB, b);

	#ifdef PRINT
	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles to do binExtEuclid was %d\n", time_spent);
	}
	#endif


	unsigned long long js = oneMS;

	//beginModul = clock();
	//Convert js to montgomery space
	//js = modul64(js, 0, b);
	//Convert newKB back into Montgomery space
	newKB = modul64(newKB, 0, b);
	//endModul = clock();
	//time_spent = (endModul - beginModul);
	//modultime += time_spent;
	//modul = modul + 2;


	//Do a dry run through the baby steps to find the free positions in the hash table
	#ifdef PRINT
	begin = clock();
	#endif

	unsigned int hash = 0;

	for (int j = 0; j < m; j++) {

		hash = (int)js & (m - 1);

		//if ((bits[Sints + (hash / 32)] & (1 << (hash & 31))) == 0) {
		//hashKeys[(Sm + hash1)] = (js & 0xFFFF0000) + 0x0000FFFF;
		//	bits[Sints + (hash / 32)] += (1 << (hash & 31));
		//}
		bits[Sints + (hash >> 5)] |= (1 << (hash & 31));

		js = montmul(js, newKB, mlo, mhi, mprimelo, mprimehi);
		//js = montmul64(js, newKB, b, bprime);

	}

	#ifdef PRINT
	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles doing dry run of new hash table was %d (%d inserts (baby steps) @ %d cycles average)\n", time_spent, m, time_spent / m);
	}
	#endif

	//Try to populate our new hash table array ------------------------------------------------------------
	//New method - pre-populate with non-collision elements. Backfill the spaces.
	js = oneMS;
	int lookups = 0;
	//int hash = 0;
	int store = 0;
	int key = 0;
	int pointer = 0;
	int firstFree = 0; //The first memory cell that doesn't head a linked list
	//int bitArrayCounter = 0;

	for (int j = 0; j < m; j++) {

		hash = (int)js & (m - 1);
		store = (js & 0xFFFF0000) + 0x0000FFFF; //This blanks off the last 16 bits and adds a null pointer. This will contain our pointer

		key = hashKeys[(Sm + hash)];

		//if (((bits[Sints + (hash / 32)] >> (hash&31)) & 1) == 1) {
		//if (key == 0) {
		//You were the element stored here, so subtract the 1
		//store = store;
		//bits[Sints + (hash / 32)] -= (1 << (hash & 31));
		//}

		//else {
		if (key != 0) {
			//You were a collision into this bucket. Find somewhere to live, and update the pointer
			//key = hashKeys[(Sm + hash)];

			//while ((hashKeys[Sm + firstFree] != 0) || (((bits[Sints + (firstFree / 32)] >> (firstFree & 31)) & 1) == 1)) {
			while (((bits[Sints + (firstFree >> 5)] >> (firstFree & 31)) & 1) == 1) {
				firstFree++;
			}
			//Could we do this with bfind instead?

			//for (int i = bitArrayCounter; i < (m >> 5); i++) {
			//	if (bits[Sints + i] != 0xFFFFFFFF) {
					//Find the position of the first zero, and turn it into a 1
			//		int pos = __clz(bits[Sints + i]);

			//		break;
			//	}
			//	bitArrayCounter = i;
			//}

			hashKeys[(Sm + firstFree)] = ((store - 0x0000FFFF) + (key & 0x0000FFFF)); //Store this new data, with the pointer from the head. We're now 2nd in this linked list
			store = (key & 0xFFFF0000) + firstFree + m;
			firstFree++;

		}

		hashKeys[(Sm + hash)] = store; //Update the linked list head, either with new data and a null pointer, or an updated pointer

		js = montmul(js, newKB, mlo, mhi, mprimelo, mprimehi);
		//js = montmul64(js, newKB, b, bprime);


	}


	//Finished calculating the hash table --------------------------------------------------------------------

	////Try to populate our new hash table array ------------------------------------------------------------
	////New method - first m buckets (0 to (m-1)) are the beginning of linked lists. Buckets m to (2m-1) are for the collisions
	//int lookups = 0;
	//int hash = 0;
	//int firstFree = m; //The first memory cell that doesn't head a linked list
	//int store = 0;
	//int key = 0;
	//int pointer = 0;

	//for (int j = 0; j<m; j++) {

	//	hash = js & (m - 1);
	//	store = js & 0xFFFF0000; //This blanks off the last 16 bits. This will contain our pointer

	//							 //beginLoop = clock();

	//							 //beginInsert = clock();

	//							 //int key = hashKeys[(Sm + hash)];

	//	if ((bits[Sints + (hash / 32)] & (1 << (hash & 31))) == 0) {
	//		//if (set == 0) {
	//		//if (key == 0) {
	//		//This linked list contains nothing yet, so add the element, and a zero pointer
	//		key = store;
	//		bits[Sints + (hash / 32)] += 1 << (hash & 31);
	//	}

	//	//if (set != 0) {
	//	else {
	//		key = hashKeys[(Sm + hash)];

	//		//This linked list has at least one element in it. Copy the pointer, put that in our data.
	//		pointer = key & 0x0000FFFF; //This removes the top 16 bits which contain the data, just leaves the pointer.
	//		key = (key & 0xFFFF0000) + firstFree; //Update the original data with the new pointer to this data

	//											  //We could gather these up in shared memory and write them out every so often. 
	//		hashKeys[(Sm + firstFree)] = (store + pointer); //Store this new data, with the pointer from the head. We're now 2nd in this linked list

	//		firstFree++; //Update the location of next free memory cell
	//	}


	//	//beginInsert = clock();
	//	hashKeys[(Sm + hash)] = key;
	//	//endInsert = clock();
	//	//time_spent = (endInsert - beginInsert);
	//	//inserttime += time_spent;
	//	//inserts++;

	//	//beginMont = clock();
	//	js = montmul(js, newKB, b, bprime);
	//	//endMont = clock();
	//	//time_spent = (endMont - beginMont);
	//	//montmultime += time_spent;
	//	//montmultime1 += time_spent;

	//	//montmuls++;
	//	//montmuls1++;

	//}


	////Finished calculating the hash table --------------------------------------------------------------------

	#ifdef PRINT
	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles calculating new hash table was %d (%d inserts (baby steps) @ %d cycles average)\n", time_spent, m, time_spent / m);
	}
	#endif

	begin = clock();
	//Compute KernelBase^-m (mod b)
	//unsigned long long c1 = modul64(newKB2, 0, b);
	//modul++;

	//c1  should be KernelBase^Q^-1 (mod b) computed earlier
	//Now repeatedly square it as m is a power of two

	for (int t = 0; t < shift; t++) {
		//beginMont = clock();
		c1 = montmul(c1, c1, mlo, mhi, mprimelo, mprimehi);
		//c1 = montmul64(c1, c1, b, bprime);
		//endMont = clock();
		//time_spent = (endMont - beginMont);
		//montmultime += time_spent;
		//montmuls++;
	}

	int output = -1;

	//int tMin = tMin[0];
	//int tMax = tMax[0];

	#ifdef PRINT
	if (printer) {
		printf("tMin = %d. tMax = %d\n", tMin[0], tMax[0]);
	}
	#endif

	lookups = 0;
	int countmuls = tMin[0];
	int giant = 0;
	//	int collisions = 0;

	int maxProbe = 0;
	//	float avgProbe = 0;

	unsigned long long fixedBeta = 0;
	unsigned long long beta = oneMS;

	for (int t = 0; t < tMin[0]; t++) {
		//beginMont = clock();
		beta = montmul(beta, c1, mlo, mhi, mprimelo, mprimehi);
		//beta = montmul64(beta, c1, b, bprime);
		//endMont = clock();
		//time_spent = (endMont - beginMont);
		//montmultime += time_spent;
		//montmuls++;
	}

	int leg1;
	int leg2fixed = legendre(base[0], b);
	int leg2;
	int leg = 1;

	int probe = 0;

	//int hits = 0;

	int thisk = 0;
	int corek = 0;
	int lastk = 0;
	int remainder = 0;

	//Work through the matrix of kn values
	for (int k = 0; k < *minSubs; k++) {
		//while (k < *minSubs) {
			lastk = thisk;
			thisk = knmatrix[k* *rowOffset];
			if (thisk != lastk) {
				beginLeg = clock();
				leg1 = legendre(thisk, b);
				leg2 = !(leg1^leg2fixed); //Rather than use k*base, use the multiplicative property of legendre to save any overflows
#ifdef PRINT
				leg = leg + 1;
				endLeg = clock();
				time_spent = (endLeg - beginLeg);
				legtime += time_spent;
#endif		
				fixedBeta = modul64(thisk, 0, b);
				fixedBeta = montmul(fixedBeta, beta, mlo, mhi, mprimelo, mprimehi);
				//fixedBeta = montmul64(fixedBeta, beta, b, bprime);
			}

			remainder = knmatrix[(k* *rowOffset) + 1];

			//if ((remainder % 2 == 0 && leg1 == 1) || (remainder % 2 == 1 && leg2 == 1)) {
			//	break;
			//}
			//k++;
		//}

		if ((remainder % 2 == 0 && leg1 == 1) || (remainder % 2 == 1 && leg2 == 1)) {
			//We need to do something
			unsigned long long sB = fixedBeta;
			for (int rem = 0; rem < remainder; rem++) {
				sB = montmul(sB, KernelBase, mlo, mhi, mprimelo, mprimehi);
				//sB = montmul64(sB, KernelBase, b, bprime);
			}

			for (int t = tMin[0]; t < tMax[0]; t++) {
				giant++;

				//Check if beta is in js
				//hash = sB & (m - 1);

				probe = 0;
				pointer = (int)sB & (m - 1);

				//while (true) {

				//	//This was quicker with the bit array in the past - it now appears to be faster without using the bit array
				//	int key = hashKeys[(Sm + pointer)];
				//	lookups++;

				//	probe++;
				//	if (probe > maxProbe) {
				//		maxProbe = probe;
				//	}

				//	pointer = key & 0x0000FFFF; //Remove the data, leave the pointer
				//	key = key & 0xFFFF0000; //Remove the pointer, leave the data

				//	if (((int)sB & 0xFFFF0000) == key) {

				//		unsigned long long jsnew = oneMS;

				//		for (int jval = 0; jval < m; jval++) {
				//			if (jsnew == sB) {
				//				output = t * m + jval;
				//				pointer = 0;
				//				break;
				//			}

				//			jsnew = montmul(jsnew, newKB, b, bprime);

				//		}
				//		//printf("Match in S %d. t=%d, hash=%d, probe=%d beta=%llu rem=%d. Output will be %llu | %d*%d^%d-1\n", S, t, hash, probe, beta, remainder, b, thisk, *Base, ((output*Q) + remainder));

				//	}

				//	if (pointer == 0) {
				//		break;
				//	}

				//}



				while (true) {

					//Surely this must be faster on some cards - it will save us some memory transactions!
					//if ((((bits[Sints + (hash / 32)] >> (hash & 31)) & 1) == 0) && probe == 0) {
					//	break;
					//}

					//This was quicker with the bit array in the past - it now appears to be faster without using the bit array
					key = hashKeys[(Sm + pointer)];
					lookups++;

					//if (printer && k == 6) {
					//	printf("Probe = %d. Data = %d. Pointer = %d.\n", probe, (key & 0xFFFF0000), (key & 0x0000FFFF));
					//}

					probe++;
					if (probe > maxProbe) {
						maxProbe = probe;
					}

					pointer = (key & 0x0000FFFF); //Remove the data, leave the pointer
					key = key & 0xFFFF0000; //Remove the pointer, leave the data

					if (((int)sB & 0xFFFF0000) == key) {

						js = oneMS;

						for (int jval = 0; jval < m; jval++) {
							if (js == sB) {
								output = t * m + jval;
								pointer = 0x0000FFFF;
								break;
							}

							js = montmul(js, newKB, mlo, mhi, mprimelo, mprimehi);
							//js = montmul64(js, newKB, b, bprime);

						}
						//printf("Match in S %d. t=%d, hash=%d, probe=%d beta=%llu rem=%d. Output will be %llu | %d*%d^%d-1\n", S, t, hash, probe, beta, remainder, b, thisk, *Base, ((output*Q) + remainder));

					}

					if (pointer == 0x0000FFFF) {
						break;
					}

					pointer = pointer - m;

					//if (probe > 15) {
					//printf("The pointer was %d\n", pointer);
					//	if (probe > 17) {
					//		printf("Killed by probe length\n");
					//			for (int i = 0; i < m; i++) {
					//				printf("Hash[%d] = Data: %d, Pointer: %d\n", i, (hashKeys[Sm + i]) & 0xFFFF0000, (hashKeys[Sm + i]) & 0x0000FFFF);
					//			}
					//		break;
					//	}
					//}

				}

				sB = montmul(sB, c1, mlo, mhi, mprimelo, mprimehi);
				//sB = montmul64(sB, c1, b, bprime);


			}
			if (output > 0) {
				printf("Output will be %llu | %d*%d^%d-1. Thread %d\n", b, thisk, base[0], ((output* *Q) + remainder), S);
				output = -1;
			}
		}

		//if (printer) {
		//printf("Lookups by k %d = %d\n", k, lookups);
		//}
	}

	#ifdef PRINT
	if (printer) {
		printf("Number of giant steps: %d\n", giant);
		//		printf("Number of collisions: %d\n", collisions);
		printf("Number of lookups against hash table was %d\n", lookups);
		printf("Max probe length was %d\n", maxProbe);
		//		printf("Average probe length was %f\n", avgProbe / giant);
		//printf("Number of hits was %d\n", hits);
	}

	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles to complete BSGS step was %d\n", time_spent);
		//		printf("Average (BSGS Cycles/muls) was %d\n", (time_spent / countmuls));
		printf("Average (BSGS Cycles/lookups) was %d\n", (time_spent / lookups));
	}
	#endif

	//begin = clock();

	//NOut[S] = output; //This should contain the k-value in the top 32 bits and the n-value in the low 32 bits

	//end = clock();
	//time_spent = (end - begin);
	//if (printer) {
	//	printf("Cycles to write output to NOut was %d\n", time_spent);
	//}


	time_spent = (end - beginfull);
	//NOut[S] = time_spent;
	NOut[S] = lookups;
	#ifdef PRINT
	if (printer) {
		//printf("-------------------- Creating Hash Table --------------------\n");
		//printf("Cycles doing Bit Array Lookups (creating Hash Table) was %d (%d lookups @ %d cycles average) - %d%\n", bittime, bitLookups, bittime / bitLookups, (bittime * 100 / time_spent));
		//printf("Cycles doing Bit Array Updates (creating Hash Table) was %d (%d updates @ %d cycles average) - %d%\n", bitUtime, bitUpdates, bitUtime / bitUpdates, (bitUtime * 100 / time_spent));
		//printf("Cycles doing Hash Table Inserts was %d (%d inserts @ %d cycles average) - %d%\n", inserttime, inserts, inserttime / inserts, (inserttime * 100 / time_spent));
		//printf("Cycles doing Montgomery Multiplication was %d (%d function calls @ %d cycles average) - %d%\n", montmultime1, montmuls1, montmultime1 / montmuls1, (montmultime1 * 100 / time_spent));
		//printf("Total of components to create hash table is %d cycles\n", bittime+bitUtime+inserttime+montmultime1);
		//printf("--------------- Finished Creating Hash Table ----------------\n");
		printf("Cycles doing Legendre was %d (%d function calls @ %d cycles average) - %d%\n", legtime, leg, legtime / leg, (legtime * 100 / time_spent));
		//printf("Cycles doing Modul64 was %d (%d function calls @ %d cycles average) - %d%\n", modultime, modul, modultime / modul, (modultime * 100 / time_spent));
		//printf("Cycles doing Mulul64 was %d (%d function calls @ %d cycles average) - %d%\n", mulultime, mulul, mulultime / mulul, (mulultime * 100 / time_spent));
		//printf("Cycles doing Montgomery Multiplication was %d (%d function calls @ %d cycles average) - %d%\n", montmultime, montmuls, montmultime / montmuls, (montmultime * 100 / time_spent));
		printf("Cycles to execute one full thread was %d\n", time_spent);
	}
	#endif


}



int main(int argc, char* argv[])
{
	//Deal with command line arguments using getopt
	int inp;
	int threadScale = 1;
	int blockScale = 1;
	char *abcdFile = "sr_745.abcd";

	unsigned long long low = 0;
	unsigned long long high = 0;
	int Qin = 0;
	int hashTableSize = 1;


	while ((inp = getopt(argc, argv, "b:hi:m:p:P:Q:t:")) != -1) {
		switch (inp) {
		case 'b':
			//Get the blockScale argument
			blockScale = strtol(optarg, NULL, 0);
			break;

		case 'h':
			//Print the help 
			cout << endl;
			cout << "CUDA Riesel Sieve 0.1.0 -- A sieve for multiple sequences of the form k*b^n-1" << endl;
			cout << endl;
			cout << "-i FILE  : Read in abcd sieve file called FILE." << endl;
			cout << "-m m     : Use m elements for the hash table." << endl;
			cout << "-p P0    : Start sieveing from P0. Must have a corresponding P1." << endl;
			cout << "-P P1    : Finish sieving at P1. Must have a corresponding P0. If no argument will use a scaled default for testing." << endl;
			cout << "             If no -p and -P arguments then we will default to 102297149770000 for testing." << endl;
			cout << "-Q Q     : Override subsequence value Q. Sieve k*b^n-1 as (k*b^d)*(b^Q)^m-1." << endl;
			cout << "-b SCALE : Scale the number of CUDA blocks per kernel by the integer argument SCALE." << endl;
			cout << "-t SCALE : Scale the number of CUDA threads per block by the integer argument SCALE." << endl;
			cout << "             Note that these no longer require being powers of 2. We use the formula 1<<SCALE to ensure power of 2." << endl;
			cout << "-h       : Prints this help" << endl;
			return 0;

		case 'i':
			//Get the input file argument
			abcdFile = optarg;
			//Check this is a valid filename at some point
			break;

		case 'm':
			hashTableSize = strtoull(optarg, NULL, 0);
			//Check this is large enough at some point. Can we still have problems if it is too small?
			break;

		case 'p':
			low = strtoull(optarg, NULL, 0);
			//Check this is large enough at some point. Can we still have problems if it is too small?
			break;

		case 'P':
			high = strtoull(optarg, NULL, 0);
			break;

		case 'Q':
			Qin = strtol(optarg, NULL, 0);
			if (Qin < 1 || Qin % 2 == 1) {
				cout << "Bad input parameter : Q must be an even integer greater than 0." << endl;
				return 1;
			}
			break;

		case 't':
			//Get the threadScale argument
			threadScale = strtol(optarg, NULL, 0);
			break;

		default:
			return 0;

		}
	}

	//Have a look at these limits in the future. We probably want to limit them more than this!
	if (blockScale < 1 || blockScale > 31) {
		cout << "Bad input parameter : 1 < BlockScale < 32. BlockScale is 1 by default, and should be a positive integer." << endl;
		return 1;
	}

	if (threadScale < 1 || threadScale > 31) {
		cout << "Bad input parameter : 1 < ThreadScale < 32. ThreadScale is 1 by default, and should be a positive integer." << endl;
		return 1;
	}

	if (hashTableSize < 1) {
		cout << "Bad input parameter. HashTableSize is 1 by default, and should be a positive integer." << endl;
		return 1;
	}

	cout << "Trying to launch with BlockScale = " << blockScale << " and ThreadScale = " << threadScale << endl;

	blockScale = 1 << blockScale - 1;
	threadScale = 1 << threadScale - 1;


	if (low == 0 && high == 0) {
		//Parameters were not set on the command line. Use our usual values for testing.

		//unsigned long long low = 600000;
		//unsigned long long high = 2500000000;

		//unsigned long long low = 6000000000;
		//unsigned long long high = 6004000000;

		//unsigned long long low = 1000067500000;
		//unsigned long long high = 1000070000000;

		//unsigned long long low = 1000099000000;
		//unsigned long long high = 1000100000000;

		//unsigned long long low = 102254819500000L;
		low = 102297149770000L;
		unsigned long long diff = 2500000L;
		high = low + (diff*threadScale*blockScale);
		//unsigned long long high = 102297160000000L;
	}

	if (high < low) {
		cout << "Bad input parameters : P1 < P0. We want to sieve primes p in the range P0 <= p <= P1." << endl;
		return 1;
	}


	const int blocks = 128 * blockScale;
	const int threads = 128 * threadScale;
	const int arraySize = blocks * threads;
	const int testArraySize = arraySize * 24;
	const int hashScaling = 1;


	const int kb = 1024;
	const int mb = kb * kb;

	wcout << "CUDA version:   v" << CUDART_VERSION << endl;

	int devCount;
	hipGetDeviceCount(&devCount);

	if (devCount < 1) {
		cout << "No CUDA enabled GPU was detected" << endl;
		return 1;
	}

	wcout << "CUDA Devices: " << devCount << endl;

	hipDeviceProp_t props;

	for (int i = 0; i < devCount; ++i)
	{
		hipGetDeviceProperties(&props, i);
		wcout << props.name << ":" << endl;
		wcout << "  CC: " << props.major << "." << props.minor << endl;
		wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
		wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
		wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
		wcout << "  Block registers: " << props.regsPerBlock << endl << endl;

		wcout << "  Warp size:         " << props.warpSize << endl;
		wcout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
		wcout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << " ]" << endl;
		wcout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1] << ", " << props.maxGridSize[2] << " ]" << endl;
		wcout << "  L2 Cache Size: " << props.l2CacheSize / kb << "kb" << endl;
		wcout << endl;
	}

	if (hashTableSize <= 1) {
		unsigned long long targetHashSize = 1;
		while (targetHashSize < props.totalGlobalMem / 2) {
			targetHashSize = targetHashSize << 1;
		}
		wcout << "Should target a " << targetHashSize / mb << "mb hash table" << endl << endl;


		//Use targetHashSize to set up the hash table - int = 32 bits = 4 bytes, so divide by 4
		//Each thread requires the a hash table, so also divide by arraySize
		long long longhashTableSize = (((targetHashSize / 4) / arraySize) / hashScaling);
		hashTableSize = longhashTableSize;
	}
	else {
		hashTableSize = 2 << hashTableSize;
	}

	cout << "Each thread should have " << hashTableSize * hashScaling << " buckets, to store " << hashTableSize << " elements. (Density 1/" << hashScaling << ")" << endl;



	//Read in an ABCD file and parse ----------------------------------------------------------------------------
	string line;
	int total = 0;
	//string abcdFile = "C:\\Users\\Rob\\Documents\\Visual Studio 2015\\Projects\\CPU Sieve\\sr_108.abcd";
	//string abcdFile = "C:\\Users\\Rob\\Desktop\\TestSieve\\sr_745.abcd";
	//string abcdFile = "sr_745.abcd";

	//First pass through the ABCD file to find the number of k's and max number of n's
	int count1 = 0; //Number of k's
	int count3 = 0; //Total number of lines
					//ifstream myfile(abcdFile);
					//if (myfile.is_open())
					//{
					//	while (getline(myfile, line))
					//	{
					//		count3++;

					//			string::size_type n = line.find(" ");
					//		string token = line.substr(0, n);
					//cout << token << endl;

					//If tokens[0] == "ABCD" then this defines a new k, otherwise it is a number
					//		if (token.compare("ABCD") == 0) {
					//			count1++;
					//cout << "We're here!" << endl;
					//		}
					//	}
					//	myfile.close();
					//}

					//else cout << "Unable to open file first time" << endl;

	countKs(abcdFile, count1, count3);

	//Second pass through the ABCD file to write the values into the matrix
	//Store the k and n values in this array
	//boost::numeric::ublas::matrix<int> kns(count1, max);
	//std::list<int> kns;
	count3 = count3 + (2 * count1);
	int *kns = (int *)malloc(count3 * sizeof(int));
	int *ks = (int *)malloc(count1 * sizeof(int));

	int minN = INT_MAX;
	int maxN = 0;

	//Reset the counts
	count1 = 0;
	count3 = 0;
	int baseCPU = 0;
	ifstream myfile2(abcdFile);
	if (myfile2.is_open())
	{
		while (getline(myfile2, line))
		{
			//Tokenise the string - if the first element of the string is "ABCD" then this is a new k-value
			string::size_type n = line.find(" ");
			string token = line.substr(0, n);
			//If tokens[0] == "ABCD" then this defines a new k, otherwise it is a number
			if (token.compare("ABCD") == 0) {
				cout << "We've found ABCD. Get the k-value" << endl;
				//Insert a 0 into kns before we insert the k-value
				kns[count3] = 0;
				count3++;
				//Get the k value
				token = line.substr(n + 1);
				//cout << token << endl;
				n = token.find("*");
				string tok = token.substr(0, n);
				//cout << tok << endl;

				int kval = stoi(tok);
				kns[count3] = kval;
				count3++;
				ks[count1] = kval;
				count1++;
				//Get the base
				if (baseCPU == 0) {
					token = token.substr(n + 1);
					n = token.find("^");
					string b = token.substr(0, n);
					//cout << b << endl;
					baseCPU = stoi(b);
					cout << "The base is " << base << endl;
				}
				//Get the starting n-value - remove the square brakets
				n = token.find("[");
				token = token.substr(n + 1);
				n = token.find("]");
				token = token.substr(0, n);
				//cout << token << endl;
				total = stoi(token);
				if (total < minN) {
					minN = total;
				}
				kns[count3] = total;
				count3++;
				cout << "This is a new k-value with value " << kval << " and initial n-value " << total << endl;
			}
			else {
				//This is a number, n-value offset
				//cout << token << endl;
				int offset = stoi(token);
				total = total + offset;
				if (total > maxN) {
					maxN = total;
				}
				kns[count3] = total;
				count3++;
				//cout << count3 << endl;
			}

		}

		myfile2.close();
	}

	else cout << "Unable to open file second time" << endl;

	//End of reading ABCD file ----------------------------------------------------------------------------------
	cout << "End of reading ABCD file" << endl;
	//-----------------------------------------------------------------------------------------------------------

	//Find the optimal Q value, and use subsequences with base b^Q ----------------------------------------------
	cout << "MinN = " << minN << " and MaxN = " << maxN << endl;
	cout << "Find the optimal(ish) Q value for subsequences in base b^Q" << endl;
	int range = maxN - minN;
	int minWork = count1 * range;
	int minSubs = count1;
	int minRange = range;
	int minQ = 1;
	cout << count1 << " k-values over a range of " << range << " n-values" << endl;

	int Qlow = 2;
	int Qhigh = 24;

	if (Qin != 0) {
		//Override these values for the look below
		Qlow = Qin;
		Qhigh = Qin + 1;
	}

	//Iterate through Q values - from 2 to 24(?)
	//At the moment, Q must be a multiple of 2 to work with the quadratic residues
	for (int Q = Qlow; Q < Qhigh; Q = Q + 2) {
		int subsequences = 0;
		//Count the number of subsequences
		bool *subseq = (bool *)malloc(count1*Q * sizeof(bool));
		memset(subseq, false, count1*Q * sizeof(bool));
		int whichk = -1;
		for (int qq = 0; qq < count3; qq++) {
			if (kns[qq] == 0) {
				whichk++;
				//Skip the k-value itself which is next in the array
				qq++;
			}
			else {
				//This is an n-value
				int mod = kns[qq] % Q;
				subseq[whichk*Q + mod] = true;
			}
		}
		//Count the number of true's in subseq - this is our number of subsequences if we use Q
		for (int qqq = 0; qqq < count1*Q; qqq++) {
			if (subseq[qqq] == true) {
				subsequences++;
			}
		}

		range = (maxN / Q) - (minN / Q) + 1;
		if (subsequences*range < minWork) {
			minQ = Q;
			minWork = subsequences * range;
			minSubs = subsequences;
			minRange = range;
		}
		cout << "Q=" << Q << ". Work= " << minWork << ". Range = " << range << ". Subsequences= " << subsequences << endl;
		free(subseq);
	}

	//Successfully found Q --------------------------------------------------------------------------------------
	if (Qin == 0) {
		cout << "Min Work has Q=" << minQ << ". Work= " << minWork << ". Range = " << minRange << ". Subsequences= " << minSubs << endl;
	}

	//Now we have minQ, we can calculate tMin and tMax to send to the gpu
	int shift = -1;
	int temp = hashTableSize;
	while (temp) {
		temp = temp >> 1;
		shift++;
	}
	int tMinCPU = (minN / minQ) >> shift;
	int tMaxCPU = (((maxN / minQ) + 1) >> shift) + 1;

	cout << "CPU thinks tMin = " << tMinCPU << " and tMax = " << tMaxCPU << endl;

	//Create a bit array that will tell us which n-values are interesting for each subsequence
	//minQ is the Q we will use, minSubs is the number of subsequences (rows in this matrix). range/32 is the number of ints to represent the range.
	int qRange = (minRange / 32) + 1;
	int rowoffset = qRange + 2;
	int *matrix = (int *)calloc(minSubs * rowoffset, sizeof(int));
	//The first column will contain the k, the second column will contain the remainder
	//Work through kns - if we find a new remainder add it to the next row
	int minimum = minN / minQ;
	cout << "Minimum = " << minimum << endl;
	int ak = 0;
	for (int qq = 0; qq < count3; qq++) {
		if (kns[qq] == 0) {
			qq++;
			ak = kns[qq];
		}
		else {
			//This is an n-value
			int n = kns[qq];
			int mod = n % minQ;
			//Check to see if this remainder is already in the matrix - if not add a new row for it

			//n should be rewritten in the form n=Qm+r. So subtract r and divide by Q to get m
			int m = ((n - mod) / minQ) - minimum;
			//Divide by 32 to find the correct bit, and do 31 - m (mod 32) to find the position
			int bit = m / 32;
			int pos = 31 - (m & 31);

			bool added = false;

			for (int rows = 0; rows < minSubs; rows++) {
				if ((matrix[rows*rowoffset] == ak) && (matrix[(rows*rowoffset) + 1] == mod)) {
					//This row has already been used, so set the correct bit to 1
					matrix[(rows*rowoffset) + bit + 2] += 1 << pos;
					added = true;
				}
			}
			if (added == false) {
				//The row didn't exist. Create it and add this element.
				for (int rows = 0; rows < minSubs; rows++) {
					if (matrix[rows*rowoffset] == 0) {
						//This row is empty
						matrix[rows*rowoffset] = ak;
						matrix[(rows*rowoffset) + 1] = mod;
						matrix[(rows*rowoffset) + bit + 2] += 1 << pos;
						break;
					}
				}
			}
		}
	}

	//Print out the bit array
	//for (int rows = 0; rows <1; rows++) {
	//cout << matrix[rows*rowoffset] << endl;
	//cout << matrix[(rows*rowoffset) + 1] << endl;
	//cout << matrix[(rows*rowoffset) + 2] << endl;
	//cout << matrix[(rows*rowoffset) + 3] << endl;
	//cout << matrix[(rows*rowoffset) + 4] << endl;
	//}


	//Generate Primes -------------------------------------------------------------------------------------------

	//unsigned long long *KernelP = (unsigned long long *)malloc(arraySize * sizeof(unsigned long long));
	int *NOut;
	unsigned long long *KernelP;
	//int *NOut = (int *)malloc(arraySize * sizeof(int));
	hipHostMalloc((void **)&NOut, arraySize * sizeof(int));
	hipHostMalloc((void **)&KernelP, arraySize * sizeof(unsigned long long));

	//Low should be greater than the primes we use below.


	unsigned long long startLow = low; //Don't touch this. Used for timing purposes

	//Use the idea of a segmented sieve. Generate a list of small primes first
	//If we use too many small primes then it can affect sieving, i.e. we cant use small -p values. Check for this in future
	clock_t begin = clock();
	int primeCount = 16384 * 128;
	int count = 0;
	bool *primes = (bool *)malloc(primeCount * 24 * sizeof(bool));
	unsigned int *smallP = (unsigned int *)malloc(primeCount * sizeof(unsigned int));
	memset(primes, true, primeCount * 24 * sizeof(bool));

	//First candidate will be 3, followed by each odd number in turn
	for (unsigned int i = 3; i < INT32_MAX; i += 2) {
		if (primes[i] == true) {
			smallP[count] = i;

			//Update count, and check to see if we are full
			count++;
			if (count == primeCount) {
				break;
			}

			//Now mark off all multiples of this in the boolean array
			for (int j = i * 2; j < primeCount * 24; j += i) {
				primes[j] = false;
			}
		}
	}
	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	cout << "Time generating small primes " << time_spent << "s" << endl;


	unsigned int *mark1 = (unsigned int *)malloc(((testArraySize / 32) + 1) * sizeof(unsigned int));


	//Try setting up the GPU just once

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (one output, seven input). 
	//Give all vectors same size for now, we can change this afterwards

	cudaStatus = hipMalloc((void**)&dev_a, arraySize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, arraySize * sizeof(unsigned long long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_c, minSubs * rowoffset * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//cudaStatus = hipMalloc((void**)&dev_e, sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//	goto Error;
	//}

	cudaStatus = hipMalloc((void**)&dev_f, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_g, arraySize * hashTableSize * hashScaling * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_h, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_i, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_j, ((arraySize * hashTableSize) / 32) * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_k, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//cudaStatus = hipMalloc((void**)&tMin, sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//	goto Error;
	//}

	//cudaStatus = hipMalloc((void**)&tMax, sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//	goto Error;
	//}

	cudaStatus = hipMalloc((void**)&dev_n, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//Copy the data to the correct GPU buffers

	//Lets try storing the k values and remainders in constant memory instead
	//cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(dev_c), ks2, (count1 * 3 + minSubs + 3) * sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy to constant memory failed!");
	//	cout << (count1 * 3 + minSubs + 3) * sizeof(int) << "bytes" << endl;
	//	goto Error;
	//}
	cudaStatus = hipMemcpy(dev_c, matrix, minSubs * rowoffset * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		//fprintf(stderr, "hipMemcpy to constant memory failed!");
		cout << (count1 * 3 + minSubs + 3) * sizeof(int) << "bytes" << endl;
		goto Error;
	}

	//cudaStatus = hipMemcpy(dev_e, &base, sizeof(int), hipMemcpyHostToDevice);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy failed!");
	//	goto Error;
	//}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(base), &baseCPU, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy to constant memory failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_f, &rowoffset, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_h, &hashTableSize, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_i, &hashScaling, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_k, &minQ, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//cudaStatus = hipMemcpy(tMin, &tMin, sizeof(int), hipMemcpyHostToDevice); //minN
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy failed!");
	//	goto Error;
	//}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(tMin), &tMinCPU, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy to constant memory failed!");
		goto Error;
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(tMax), &tMaxCPU, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy to constant memory failed!");
		goto Error;
	}

	//cudaStatus = hipMemcpy(tMax, &tMax, sizeof(int), hipMemcpyHostToDevice); //maxN
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy failed!");
	//	goto Error;
	//}

	cudaStatus = hipMemcpy(dev_n, &minSubs, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	//cudaStatus = hipMemcpy(dev_j, bits, ((arraySize * hashTableSize * hashScaling) / 32) * sizeof(unsigned int), hipMemcpyHostToDevice);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy failed!");
	//	goto Error;
	//}

	//Create the first set of primes for the GPU before we start
	if (low % 2 == 0) {
		//Make sure low is odd. Go back by one if necessary
		low = low - 1;
		cout << "We've reduced low by 1 to make it odd" << endl;
	}

	cout << "Low is now set to " << low << endl;
	generateGPUPrimes(KernelP, low, smallP, testArraySize, primeCount, arraySize, mark1);

	hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	int kernelCount = 0;
	clock_t loopTime = clock();
	//From here we need to loop to keep the GPU busy. 
	while (low < high) {

		kernelCount++;
		cout << "Executing kernel number " << kernelCount << endl;

		unsigned long long minPrime = KernelP[0];
		unsigned long long maxPrime = KernelP[arraySize - 1];
		unsigned long long progress = maxPrime - minPrime;

		cout << "Min Prime = " << minPrime << ". Max Prime = " << maxPrime << ". Progress = " << progress << endl;
		cout << "Array Size = " << arraySize << endl;

		//Set low to the next odd number above maxPrime. The CPU will generate the netx batch of primes while the GPU is working
		low = maxPrime + 2;


		begin = clock();
		cout << "Try to launch the CUDA kernel" << endl;

		hipStream_t stream0;
		//hipStream_t stream1;
		hipStreamCreate(&stream0);
		//hipStreamCreate(&stream1);

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpyAsync(dev_b, KernelP, arraySize * sizeof(unsigned long long), hipMemcpyHostToDevice, stream0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy input failed!");
		}

		cudaStatus = hipMemsetAsync(dev_g, 0, arraySize * hashTableSize * hashScaling * sizeof(int), stream0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy input failed!");
		}

		cudaStatus = hipMemsetAsync(dev_j, 0, ((arraySize * hashTableSize) / 32) * sizeof(unsigned int), stream0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy input failed!");
		}

		hipStreamSynchronize(stream0);

		//hipEvent_t start, stop;
		//hipEventCreate(&start);
		//hipEventCreate(&stop);
		// Launch a kernel on the GPU with one thread for each element.
		//hipEventRecord(start);


		addKernel1 << <blocks, threads, 0, stream0 >> >(dev_a, dev_b, dev_c, dev_f, dev_g, dev_h, dev_i, dev_j, dev_k, dev_n);
		//addKernel1 << <blocks, threads, 0, stream1 >> >(dev_a, dev_b, dev_c, dev_e, dev_f, dev_g, dev_h, dev_i, dev_j);

		//This uses too much shared memory and kills occupancy. Really we want to use no more than 16 ints per thread (for 64 threads per block)!
		//addKernel1 << <blocks, threads, ((threads*hashElements*hashDensity) / 32)*sizeof(int), stream0 >> >(dev_a, dev_b, dev_c, dev_e, dev_f, dev_g, dev_h, dev_i);

		//addKernel1<<<blocks,threads,0,stream1>>>(dev_a, dev_b, dev_c, dev_e, dev_f, dev_g, dev_h);
		//hipEventRecord(stop);

		//hipEventSynchronize(stop);
		//float milliseconds = 0;
		//hipEventElapsedTime(&milliseconds, start, stop);
		//printf("Time taken: %f ms \n", milliseconds);

		// Check for any errors launching the kernel
		//cudaStatus = hipGetLastError();
		//if (cudaStatus != hipSuccess) {
		//    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		//}

		//We should try to generate the next array of primes in here!
		generateGPUPrimes(KernelP, low, smallP, testArraySize, primeCount, arraySize, mark1);

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpyAsync(NOut, dev_a, arraySize * sizeof(int), hipMemcpyDeviceToHost, stream0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy output failed!");
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			hipGetLastError();
		}


		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		cout << "Time to execute kernel (outside function) " << time_spent << "s" << endl;

		printf("%d,%d,%d,%d,%d,%d,%d,%d,%d,%d\n",
			NOut[0], NOut[32], NOut[64], NOut[96], NOut[128], NOut[160], NOut[192], NOut[224], NOut[256], NOut[288]);

		int maxCycles = 0;
		int maxThreadID = 0;
		int minCycles = INT_MAX;
		int minThreadID = 0;
		for (int i = 0; i < arraySize; i++) {
			if (NOut[i] > maxCycles) {
				maxCycles = NOut[i];
				maxThreadID = i;
			}
			if (NOut[i] < minCycles) {
				minCycles = NOut[i];
				minThreadID = i;
			}
		}

		cout << "Max Cycles in this kernel was " << maxCycles << " in Thread " << maxThreadID << endl;
		cout << "Min Cycles in this kernel was " << minCycles << " in Thread " << minThreadID << endl;

		long long totalCycles = 0;
		for (int i = 0; i < arraySize; i++) {
			totalCycles += NOut[i];
		}

		cout << "Total cycles for this kernel was " << totalCycles << " @ " << totalCycles / (arraySize) << " cycles average" << endl << endl;

	}

	clock_t loopEnd = clock();
	time_spent = (double)(loopEnd - loopTime) / CLOCKS_PER_SEC;
	cout << "Time taken " << time_spent << "s" << endl;
	cout << "Time per kernel " << time_spent / kernelCount << endl;
	cout << "Progress = " << (low - 2) - startLow << " at " << ((low - 2) - startLow) / time_spent << " p/sec" << endl << endl;

	//Reprint the CUDA info
	wcout << "CUDA version:   v" << CUDART_VERSION << endl;

	hipGetDeviceCount(&devCount);
	wcout << "CUDA Devices: " << devCount << endl;

	for (int i = 0; i < devCount; ++i)
	{
		hipGetDeviceProperties(&props, i);
		wcout << props.name << ":" << endl;
		wcout << "  CC: " << props.major << "." << props.minor << endl;
		wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
		wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
		wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
		wcout << "  Block registers: " << props.regsPerBlock << endl;
		wcout << "  L2 Cache Size: " << props.l2CacheSize / kb << "kb" << endl;
		wcout << endl;
	}

	cout << "Each thread used " << hashTableSize * hashScaling << " buckets, to store " << hashTableSize << " elements. (Density 1/" << hashScaling << ")" << endl;
	cout << "Hash table size was " << (hashTableSize*hashScaling * 4 * arraySize) / mb << "mb of GPU RAM" << endl;
	cout << "Blocksize = " << blocks << ". Threads per block = " << threads << "." << endl;

Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	//hipFree(dev_e);
	hipFree(dev_f);
	hipFree(dev_g);
	hipFree(dev_h);
	hipFree(dev_i);
	hipFree(dev_j);
	//return cudaStatus;

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

void generateGPUPrimes(unsigned long long *KernelP, unsigned long long low, unsigned int *smallP, int testArraySize, int primeCount, int arraySize, unsigned int *mark1) {
	clock_t begin = clock();
	//unsigned int *mark1 = (unsigned int *)malloc(((testArraySize / 32) + 1)*sizeof(unsigned int));
	//memset(mark1, 0, ((testArraySize / 32) + 1)*sizeof(unsigned int));
	unsigned int diff = 0;

	//Lets deal with multiples of 3 separately using a mask.
	unsigned int mask[3];
	mask[0] = 0x92492492;
	mask[1] = 0x49249249;
	mask[2] = 0x24924924;

	unsigned int offset = low % 3;

	for (int i = 0; i < ((testArraySize / 32) + 1); i++) {
		mark1[i] = mask[(offset + i) % 3];
	}

	//unsigned int mask1[5];
	//mask1[0] = 0x84210842;
	//mask1[1] = 0x42108421;
	//mask1[2] = 0x21084210;
	//mask1[3] = 0x10842108;
	//mask1[4] = 0x08421084;

	//offset = low % 5;

	//for (int i = 0; i < ((testArraySize / 32) + 1); i++) {
	//	mark1[i] = mark1[i] | mask1[(offset + i) % 5];
	//}

	//1000 0100 0010 0001 0000 1000 0100 0010

	//Start at 2 now as we deal with 3 and 5 with masks
	for (int i = 1; i < primeCount; i++) {
		unsigned int smallPrime = smallP[i];
		unsigned int mod = low % smallPrime;

		if (mod == 0) {
			diff = 0;
		}
		else {
			if (mod % 2 == 1) {
				mod = mod + smallPrime;
			}
			mod = mod >> 1;
			diff = (smallPrime - mod);
		}

		for (int k = diff; k < testArraySize; k += smallPrime) {
			//mark[k] = false;
			//Take k and divide it by 32 to work out which int we are in. Shift by the remainder
			int intbool = k / 32; //Find the right int
			int intshift = k & 31; //Work out the shift
			if (((mark1[intbool] >> (31 - intshift)) & 1) == 1) {
				//Do nothing - this bit is already 1
			}
			else {
				mark1[intbool] += (0x80000000 >> intshift);
			}
		}

	}

	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	cout << "Time marking the prime array " << time_spent << "s" << endl;

	begin = clock();
	//Use mark1 to decide if we need to add a prime. We're looking for 1 entries in these ints
	int countPrimes = 0;
	for (int i = 0; i < testArraySize; i++) {
		int intbool = i / 32; //Find the right int
		int intshift = i & 31; //Work out the shift
		if (((mark1[intbool] >> (31 - intshift)) & 1) == 0) {
			KernelP[countPrimes] = 2 * i + low;
			countPrimes++;
			if (countPrimes == arraySize) {
				cout << "We got as far as " << i + low << " out of " << low + (testArraySize) << endl;
				break;
			}
		}
	}

	end = clock();
	time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	cout << "Time generating kernel primes " << time_spent << "s" << endl;

}




