#include "hip/hip_runtime.h"
//Complete - Read in an ABCD file
//Complete - CUDA code with correct outputs

//TODO - Fix inputs to the CUDA function call to be an actual ABCD file and list of primes
//TODO - Function to create list of primes


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <list>

using namespace std;

hipError_t addWithCuda(int *NOut, unsigned long long *KernelP, unsigned int size, const int blocks, const int threads, int *hashKeys, int hashTableElements);


int *dev_a = 0; //NOut
unsigned long long *dev_b = 0; //KernelP
int *dev_c = 0; //kns
int *dev_e = 0; //Base
int *dev_f = 0; //counterIn
int *dev_g = 0; //HashTable Keys
//unsigned long long *dev_h = 0; //HashTable Values
hipError_t cudaStatus;


__device__  __forceinline__ unsigned long long xbinGCD(unsigned long long a, unsigned long long b)
{
	unsigned long long alpha, beta, u, v;
	u = 1; v = 0;
	alpha = a; beta = b; // Note that alpha is
						 // even and beta is odd.
						 /* The invariant maintained from here on is:
						 2a = u*2*alpha - v*beta. */
	while (a > 0) {
		a = a >> 1;
		if ((u & 1) == 0) { // Delete a common
			u = u >> 1; v = v >> 1; // factor of 2 in
		} // u and v.
		else {
			/* We want to set u = (u + beta) >> 1, but
			that can overflow, so we use Dietz's method. */
			u = ((u ^ beta) >> 1) + (u & beta);
			v = (v >> 1) + alpha;
		}
	}
	//*pu = u;
	//*pv = v;
	return v;
}

__device__ __forceinline__ unsigned long long modul64(unsigned long long x, unsigned long long y, unsigned long long z) {
	/* Divides (x || y) by z, for 64-bit integers x, y,
	and z, giving the remainder (modulus) as the result.
	Must have x < z (to get a 64-bit result). This is
	checked for. */
	long long i, t;
	if (x >= z) {
		printf("Bad call to modul64, must have x < z.");
	}
	for (i = 1; i <= 64; i++) { // Do 64 times.
		t = (long long)x >> 63; // All 1's if x(63) = 1.
		x = (x << 1) | (y >> 63); // Shift x || y left
		y = y << 1; // one bit.
		if ((x | t) >= z) {
			x = x - z;
			y = y + 1;
		}
	}
	return x; // Quotient is y.
}

__device__ __forceinline__ unsigned long long montmul(unsigned long long abar, unsigned long long bbar, unsigned long long m, unsigned long long mprime) {
	unsigned long long thi, tlo, tm, tmmhi, tmmlo;
	unsigned long long uhi, ulo;
	unsigned int ov;
	
	//mulul64(abar, bbar, &thi, &tlo); // t = abar*bbar.
	thi = __umul64hi(abar, bbar);
	tlo = abar*bbar;
	/* Now compute u = (t + ((t*mprime) & mask)*m) >> 64.
	The mask is fixed at 2**64-1. Because it is a 64-bit
	quantity, it suffices to compute the low-order 64
	bits of t*mprime, which means we can ignore thi. */
	tm = tlo*mprime;
	//mulul64(tm, m, &tmmhi, &tmmlo); // tmm = tm*m.
	tmmhi = __umul64hi(tm, m);
	tmmlo = tm*m;
	
	//Replace this with ptx
	//ulo = tlo + tmmlo; // Add t to tmm
	//uhi = thi + tmmhi; // (128-bit add).
	//if (ulo < tlo) uhi = uhi + 1; // Allow for a carry.
	// The above addition can overflow. Detect that here.
	//ov = (uhi < thi) | ((uhi == thi) & (ulo < tlo));

	asm("add.cc.u64 %0, %3, %4;\n\t" //Add tlo and tmmlo and set carry out. 
		"addc.cc.u64 %1, %5, %6;\n\t" //Add thi and tmmhi, use the previous carry and set carry out.
		"addc.u32 %2, 0, 0;" //This sets ov to 1 if the previous addition overflowed.
		: "=l"(ulo), "=l"(uhi) "=r"(ov) : "l"(tlo), "l"(tmmlo), "l"(thi), "l"(tmmhi)
		);

	//asm("add.cc.u64 %0, %0, %3;\n\t" //Add tlo and tmmlo and set carry out. 
	//	"addc.cc.u64 %1, %1, %4;\n\t" //Add thi and tmmhi, use the previous carry and set carry out.
	//	"addc.u32 %2, 0, 0;" //This sets ov to 1 if the previous addition overflowed.
	//	: "=l"(tlo), "=l"(thi) "=r"(ov) : "l"(tmmlo), "l"(tmmhi)
	//	);

	//if (ov > 0 || thi >= m) // If u >= m,
	//	thi = thi - m; // subtract m from u.
	//return thi;



	//ulo = uhi; // Shift u right
	if (ov > 0 || uhi >= m) // If u >= m,
		uhi = uhi - m; // subtract m from u.
	return uhi;
}

__device__ __forceinline__ long long binExtEuclid(long long a, long long b) {
	long long u = b;
	long long v = a;
	long long r = 0;
	long long s = 1;
	long long x = a;
	while (v>0) {
		if ((u & 1) == 0) {
			u = u >> 1;
			if ((r & 1) == 0) {
				r = r >> 1;
			}
			else {
				r = (r + b) >> 1;
			}
		}
		else {
			if ((v & 1) == 0) {
				v = v >> 1;
				if ((s & 1) == 0) {
					s = s >> 1;
				}
				else {
					s = (s + b) >> 1;
				}
			}
			else {
				x = u - v;
				if (x>0) {
					u = x;
					r = r - s;
					if (r<0) {
						r = r + b;
					}
				}
				else {
					v = x * -1;
					s = s - r;
					if (s<0) {
						s = s + b;
					}
				}
			}
		}
	}
	if (r >= b) {
		r = r - b;
	}
	if (r<0) {
		r = r + b;
	}
	return(r);
}


__global__ void addKernel1(int *NOut, unsigned long long *KernelP, int *ks, int *Base, int *counterIn, int *hashKeys)
{
	clock_t beginfull = clock();
	clock_t begin = clock();
	int i = threadIdx.x;
	int block = blockIdx.x;
	int N = blockDim.x * gridDim.x; //This is threads*blocks
	
	//This deals with the hashTables
	int m = 512;
	int shift = 9; //m=2^shift
	int mem = m * 4; //This is hashTableElements * 4 to reduce collisions. Must be a power of 2
	int memN = (N * mem)-1; //We use this for doing cheap modulo's as N*mem should be a power of 2
	unsigned int bitArray[64]; //Bit array for hash table

	//__shared__ unsigned int sharedBA[64*32];
	//for (int ii = 0; ii < 64; ii++) {
	//	sharedBA[i + 32*ii] = 0;
	//}

	for (int ii = 0; ii < 64; ii++) {
		bitArray[ii] = 0;
	}
	
	int S = (block * blockDim.x) + i; //This is this block ID*threads in a block + threadID
	//int SiShift = S << shift;
	unsigned long long b = KernelP[S];
	unsigned long long KernelBase = *Base;
	int outputBase = KernelBase;
	int counter = *counterIn;
	int barretts = 0;

	bool printer = false;
	if (i == 0 & block == 0) {
		printer = true;
	}

	clock_t end = clock();
	int time_spent = (end - begin);
	if (printer) {
		printf("KernelBase = %d\n", KernelBase);
		printf("Cycles to complete variable setup was %d\n", time_spent);
	}

	begin = clock();

	unsigned long long bprime = xbinGCD(9223372036854775808, b);

	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles to do xbinGCD was %d\n", time_spent);
	}

	begin = clock();

	KernelBase = binExtEuclid(KernelBase, b);

	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles to do binExtEuclid was %d\n", time_spent);
	}

	begin = clock();
	//Try to populate our new hash table array ------------------------------------------------------------
	int lookups = 0;
	int hash = 0;
	unsigned long long js = 1;

	//Convert js to montgomery space
	js = modul64(js, 0, b);
	KernelBase = modul64(KernelBase, 0, b);
	//unsigned long long int jsArray[1024];
	//unsigned int keys[4096];

	//int index = 0;
	for (int j = 0; j<m; j++) {


		clock_t beginindex;
		if (printer & j == m >> 1) {
			beginindex = clock();
		}

		hash = js & mem-1;
		//hash = js & 4095;
		//int index = hash*N + S;

		if (printer & j == m >> 1) {
			clock_t endindex = clock();
			time_spent = (endindex - beginindex);
			printf("Cycles to calculate hash and index was %d\n", time_spent);
		}

		//Basic linear probing
		for (int probe = 0; probe < m; probe++) {
			lookups++;

			clock_t beginhash;
			if (printer & j == m >> 1) {
				beginhash = clock();
			}
				
			//if ((hashKeys[index]) == 0) {
			if ((bitArray[hash / 32] & (1 << (hash & 31))) == 0) {
			//if ((sharedBA[i + 32*(hash / 32)] & (1 << (hash & 31))) == 0) {
				bitArray[hash / 32] += 1 << (hash & 31);
				//sharedBA[i + 32*(hash / 32)] += 1 << (hash & 31);
				//index = hash*N + S;

				//hashKeys[(hash*N + S)] = j;
				//hashValues[SiShift + j] = js;
				//hashKeys[(hash*N + S)] = js;
				hashKeys[(S*mem + hash)] = js;
				//keys[hash] = js;
				//jsArray[j] = js;

				if (printer & j == m >> 1) {
					clock_t endhash = clock();
					time_spent = (endhash - beginhash);
					printf("Cycles to add key and value to hash table was %d\n", time_spent);
				}

				break;
			}
			
			hash = (hash+1) & (mem - 1);
			//index = index + N;
			//index = index & memN;
		}

		clock_t beginmul;
		if (printer & j == m >> 1) {
			beginmul = clock();
		}

		js = montmul(js, KernelBase, b, bprime);
		barretts++;

		if (printer & j == m >> 1) {
			clock_t endmul = clock();
			time_spent = (endmul - beginmul);
			printf("Cycles to perform a montmul was %d\n", time_spent);
		}

	}
	if (printer) {
		printf("Number of lookups while inserting into the hash table was %d\n", lookups);
	}

	//Finished calculating the hash table --------------------------------------------------------------------

	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles calculating new hash table was %d\n", time_spent);
		printf("Average was %d\n", time_spent/m);
	}


	begin = clock();
	//Compute KernelBase^-m (mod b)
	unsigned long long c1 = outputBase;

	c1 = modul64(c1, 0, b);
	
	//if (printer) {
	//	printf("Here 1\n");
	//}
	//This should be KernelBase^-1 (mod b)
	//Now repeatedly square it as m is a power of two
															 
	for (int t = 0; t<shift; t++) {
		c1 = montmul(c1, c1, b, bprime);
		barretts++;
	}

	//if (printer) {
	//	printf("Here 2\n");
	//}

	long long output = -5;
	int NMin = 20;
	int NMax = 1000;
	int tMin = NMin >> shift;
	lookups = 0;
	int countmuls = tMin;

	//int index = 0;

	//if (printer) {
	//	printf("Here 3\n");
	//}

	for (int k = 0; k<counter; k++) {
		unsigned long long beta = ks[k];
		beta = modul64(beta, 0, b);
		
		for (int t = 0; t<tMin; t++) {
			beta = montmul(beta, c1, b, bprime);
			barretts++;
		}

		//if (printer) {
		//	printf("Here 4\n");
		//}

		for (int t = tMin; t<140; t++) {

			//Check if beta is in js
			hash = beta & mem-1;
			//index = hash*N + S;

			//Its possible beta is here, use linear probing to check
			for (int probe = 0; probe < m; probe++) {
				lookups++;
				//if ((hashKeys[hash*N + S]) == 0) {
				if ((bitArray[hash / 32] & (1 << (hash & 31))) == 0) {
				//if ((sharedBA[i + 32*(hash / 32)] & (1 << (hash & 31))) == 0) {
					//Beta is not here
					break;
				}
				//else if (hashValues[(SiShift + hashKeys[index])]  == beta) {
				//else if (jsArray[hashKeys[index]] == beta) {
				//else if (hashKeys[hash*N + S] == beta) {
				else if (hashKeys[S*mem + hash] == beta) {
					//if (printer) {
					//	printf("Here 5\n");
					//}
					lookups++;
					//We've found beta
					//We've had a match
					//output = (t*m + (hashKeys[index]));
					//output = (t*m + (keys[hash]));
					//Find the j value
					unsigned long long jsnew = 1;
					jsnew = modul64(jsnew, 0, b);
					for (int jval = 0; jval < m; jval++) {
						if (jsnew == beta) {
							output = t*m + jval;
							break;
						}
						jsnew = montmul(jsnew, KernelBase, b, bprime);
						barretts++;
					}
					//printf("Match in Thread %d, Block %d. t=%d, hash=%d, probe=%d beta=%llu. Output will be %llu | %d*%d^%d-1\n", i, block, t, hash, probe, beta, b, ks[k], outputBase, output);
					if (printer) {
						printf("Here 6\n");
					}
					break;
				}
				//index = index + N;
				//index = index & memN;
				hash = (hash + 1) & (mem - 1);
			}
			
			beta = montmul(beta, c1, b, bprime);
			countmuls++;
			barretts++;
		}

		if (output < NMin) {
			output=-3;
		}
		else if (output > NMax) {
			output = -4;
		}
		else {
			printf("Output will be %llu | %d*%d^%d-1\n", b, ks[k], outputBase, output);
			output = -5;
		}

	}
	if (printer) {
		printf("Number of lookups against hash table was %d\n", lookups);
	}

	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles to complete BSGS step was %d\n", time_spent);
		printf("Average (BSGS Cycles/muls) was %d\n", (time_spent/countmuls));
		printf("Average (BSGS Cycles/lookups) was %d\n", (time_spent / lookups));
	}

	begin = clock();

	NOut[S] = output; //This should contain the k-value in the top 32 bits and the n-value in the low 32 bits

	end = clock();
	time_spent = (end - begin);
	if (printer) {
		printf("Cycles to write output to NOut was %d\n", time_spent);
	}

	if (printer) {
		printf("Total number of montgomery multiplies was %d\n", barretts);
	}

	time_spent = (end - beginfull);
	if (printer) {
		printf("Cycles to execute one full thread was %d\n", time_spent);
	}
}



int main()
{
	const int kb = 1024;
	const int mb = kb * kb;

	wcout << "CUDA version:   v" << CUDART_VERSION << endl;

	int devCount;
	hipGetDeviceCount(&devCount);
	wcout << "CUDA Devices: " << devCount << endl;

	hipDeviceProp_t props;

	for (int i = 0; i < devCount; ++i)
	{
		hipGetDeviceProperties(&props, i);
		wcout << props.name << ":" << endl;
		wcout << "  CC: " << props.major << "." << props.minor << endl;
		wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
		wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
		wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
		wcout << "  Block registers: " << props.regsPerBlock << endl << endl;

		wcout << "  Warp size:         " << props.warpSize << endl;
		wcout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
		wcout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << " ]" << endl;
		wcout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1] << ", " << props.maxGridSize[2] << " ]" << endl;
		wcout << "  L2 Cache Size: " << props.l2CacheSize / kb << "kb" << endl;
		wcout << endl;
	}

	//Read in an ABCD file and parse ----------------------------------------------------------------------------
	string line;
	int total = 0;
	//string abcdFile = "C:\\Users\\Rob\\Documents\\Visual Studio 2015\\Projects\\CPU Sieve\\sr_108.abcd";
	//string abcdFile = "C:\\Users\\Rob\\Desktop\\TestSieve\\sr_745.abcd";
	string abcdFile = "sr_745.abcd";

	//First pass through the ABCD file to find the number of k's and max number of n's
	int count1 = 0; //Number of k's
	int count3 = 0; //Total number of lines
	ifstream myfile(abcdFile);
	if (myfile.is_open())
	{
		while (getline(myfile, line))
		{
			count3++;

			string::size_type n = line.find(" ");
			string token = line.substr(0,n);
			//cout << token << endl;

			//If tokens[0] == "ABCD" then this defines a new k, otherwise it is a number
			if (token.compare("ABCD") == 0) {
				count1++;
				//cout << "We're here!" << endl;
			}
		}
		myfile.close();
	}

	else cout << "Unable to open file first time" << endl;

	//Second pass through the ABCD file to write the values into the matrix
	//Store the k and n values in this array
	//boost::numeric::ublas::matrix<int> kns(count1, max);
	//std::list<int> kns;
	count3 = count3 + (2 * count1);
	int *kns = (int *)malloc(count3*sizeof(int));
	int *ks = (int *)malloc(count1*sizeof(int));

	//Reset the counts
	count1 = 0;
	count3 = 0;
	int base = 0;
	ifstream myfile2(abcdFile);
	if (myfile2.is_open())
	{
		while (getline(myfile2, line))
		{
			//Tokenise the string - if the first element of the string is "ABCD" then this is a new k-value
			string::size_type n = line.find(" ");
			string token = line.substr(0, n);
			//If tokens[0] == "ABCD" then this defines a new k, otherwise it is a number
			if (token.compare("ABCD") == 0) {
				cout << "We've found ABCD. Get the k-value" << endl;
				//Insert a 0 into kns before we insert the k-value
				kns[count3] = 0;
				count3++;
				//Get the k value
				token = line.substr(n+1);
				//cout << token << endl;
				n = token.find("*");
				string tok = token.substr(0, n);
				//cout << tok << endl;
				
				int kval = stoi(tok);
				kns[count3] = kval;
				count3++;
				ks[count1] = kval;
				count1++;
				//Get the base
				if (base == 0) {
					token = token.substr(n+1);
					n = token.find("^");
					string b = token.substr(0, n);
					//cout << b << endl;
					base = stoi(b);
					cout << "The base is " << base << endl;
				}
				//Get the starting n-value - remove the square brakets
				n = token.find("[");
				token = token.substr(n + 1);
				n = token.find("]");
				token = token.substr(0, n);
				//cout << token << endl;
				total = stoi(token);
				kns[count3] = total;
				count3++;
				cout << "This is a new k-value with value " << kval << " and initial n-value " << total << endl;
			}
			else {
				//This is a number, n-value offset
				//cout << token << endl;
				int offset = stoi(token);
				total = total + offset;
				kns[count3] = total;
				count3++;
				//cout << count3 << endl;
			}

		}

		myfile2.close();
	}

	else cout << "Unable to open file second time" << endl;

	//End of reading ABCD file ----------------------------------------------------------------------------------
	cout << "End of reading ABCD file" << endl;


	//Generate Primes -------------------------------------------------------------------------------------------

	const int blocks = 512; 
	const int threads = 128; //These must multiply to around 65536. Larger and CUDA times out
    const int arraySize = blocks*threads;
	const int testArraySize = arraySize * 24;
	const int hashTableElements = 512;
	const int hashScaling = 4;
	unsigned long long *KernelP = (unsigned long long *)malloc(arraySize*sizeof(unsigned long long));
    //unsigned long long KernelP[arraySize] = { 0 };
	int *NOut = (int *)malloc(arraySize*sizeof(int));
    //int NOut[arraySize] = { 0 };
	int *hashKeys = (int *)malloc(arraySize * hashTableElements * hashScaling * sizeof(int));
	memset(hashKeys, 0, arraySize * hashTableElements * hashScaling * sizeof(int));
	//unsigned long long *hashValues = (unsigned long long *)malloc(arraySize * hashTableElements * sizeof(unsigned long long));
	//memset(hashValues, 0, arraySize * hashTableElements * sizeof(unsigned long long));

	//Low should be greater than the primes we use below. 
	unsigned long long low = 6000000000;
	unsigned long long high = 6004000000;

	//unsigned long long low = 1000067500000;
	//unsigned long long high = 1000070000000;

	//unsigned long long low = 1000099000000;
	//unsigned long long high = 1000100000000;

	//unsigned long long low = 600000;
	//unsigned long long high = 7000000;


	unsigned long long startLow = low; //Don't touch this. Used for timing purposes

	//Use the idea of a segmented sieve. Generate a list of small primes first
	//Could use the first 1024 primes as a starter. 8161 is the 1024th prime
	//Currently using the first 70 primes aas a starter.
	clock_t begin = clock();
	int smallPrimes = 8162;
	int primeCount = 1024;
	int s = 0;
	bool *primes = (bool *)malloc(smallPrimes*sizeof(bool));
	unsigned int *smallP = (unsigned int *)malloc(primeCount*sizeof(unsigned int));
	memset(primes, true, smallPrimes*sizeof(bool));

	int sq = smallPrimes*smallPrimes;

	for (int p = 2; p*p < sq; p++) {
		if (primes[p] == true) {
			smallP[s] = p;
			//cout << smallP[s] << endl;
			s++;
			for (int i = p*2; i < smallPrimes; i += p) {
				primes[i] = false;
			}
		}
	}
	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	cout << "Time generating small primes " << time_spent << "s" << endl;

	//Print the small primes as a check
	for (int p = 0; p < primeCount; p++) {
		//cout << smallP[p] << endl;
	}

	//Find the minimum number in [low...high] that is a multiple of primes[i]
	
	bool *mark = (bool *)malloc(testArraySize*sizeof(bool));



	//Try setting up the GPU just once

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (one output, seven input). 
	//Give all vectors same size for now, we can change this afterwards

	cudaStatus = hipMalloc((void**)&dev_a, arraySize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, arraySize * sizeof(unsigned long long));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_c, count1 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_e, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_f, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_g, arraySize * hashTableElements * hashScaling * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//cudaStatus = hipMalloc((void**)&dev_h, arraySize * hashTableElements * sizeof(unsigned long long));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMalloc failed!");
	//	goto Error;
	//}

	cudaStatus = hipMemcpy(dev_c, ks, count1 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_e, &base, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_f, &count1, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_g, hashKeys, arraySize * hashTableElements * hashScaling * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy input failed!");
		goto Error;
	}

	//cudaStatus = hipMemcpy(dev_h, hashValues, arraySize * hashTableElements * sizeof(unsigned long long), hipMemcpyHostToDevice);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy input failed!");
	//	goto Error;
	//}

	int kernelCount = 0;
	clock_t loopTime = clock();
	//From here we need to loop to keep the GPU busy. 
	while (low < high) {
		kernelCount++;
		cout << "Executing kernel number " << kernelCount << endl;
		cout << "Low is now set to " << low << endl;

		begin = clock();
		memset(mark, true, testArraySize*sizeof(bool));

		for (int i = 0; i < primeCount; i++) {
			unsigned int smallPrime = smallP[i];
			for (int j = 0; j < testArraySize; j++) {
				//if (mark[j] == true && (((low + j) % smallP[i]) == 0)) {
				if (((low + j) % smallPrime) == 0) {
					//So if low + offset can be divided by i we've found the first value divisible by i. Now mark off all i multiples
					for (int k = j; k < testArraySize; k += smallPrime) {
						mark[k] = false;
					}
					break;
				}
			}
		}
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		cout << "Time marking the prime array " << time_spent << "s" << endl;


		// Numbers which are not marked as false are prime
		begin = clock();
		int countPrimes = 0;
		for (unsigned long long i = low; i < low + (testArraySize); i++) {
			if (mark[i - low] == true) {
				KernelP[countPrimes] = i;
				countPrimes++;
				if (countPrimes == arraySize) {
					cout << "We got as far as " << i << " out of " << low + (testArraySize) << endl;
					break;
				}
				//cout << i << endl;
			}
		}
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		cout << "Time generating kernel primes " << time_spent << "s" << endl;

		unsigned long long minPrime = KernelP[0];
		unsigned long long maxPrime = KernelP[arraySize - 1];
		unsigned long long progress = maxPrime - minPrime;

		cout << "Min Prime = " << minPrime << ". Max Prime = " << maxPrime << ". Progress = " << progress << endl;
		cout << "Array Size = " << arraySize << endl;

		//End of Generating Primes ----------------------------------------------------------------------------------

		begin = clock();
		cout << "Try to launch the CUDA kernel" << endl;
		// Add vectors in parallel.
		//This uses the full ABCD file, but runs very slowly when file is big
		//hipError_t cudaStatus = addWithCuda(NOut, KernelP, kns, &base, &count3, arraySize, count3, blocks, threads);
		//This is datless - remember to change to addkernel1
		hipError_t cudaStatus = addWithCuda(NOut, KernelP, arraySize, blocks, threads, hashKeys, hashTableElements);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addWithCuda failed!");
			return 1;
		}
		end = clock();
		time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
		cout << "Time to execute kernel (outside function) " << time_spent << "s" << endl;

		printf("%d,%d,%d,%d,%d,%d,%d,%d,%d,%d\n\n",
			NOut[0], NOut[1], NOut[2], NOut[3], NOut[4], NOut[5], NOut[6], NOut[7], NOut[8], NOut[9]);

		//Set low equal to high and continue in the loop
		low = maxPrime;
	}

	clock_t loopEnd = clock();
	time_spent = (double)(loopEnd - loopTime) / CLOCKS_PER_SEC;
	cout << "Time taken " << time_spent << "s" << endl;
	cout << "Time per kernel " << time_spent / kernelCount << endl;
	cout << "Progress = " << KernelP[arraySize - 1] - startLow << " at " << (KernelP[arraySize - 1] - startLow) / time_spent << " p/sec" << endl;

	//Could also print out number of cuda cores and l2 cache size
	wcout << "L2 Cache Size: " << props.l2CacheSize / kb << "kb" << endl;

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

Error:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_e);
	hipFree(dev_f);
	hipFree(dev_g);
	//hipFree(dev_h);
	return cudaStatus;

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *NOut, unsigned long long *KernelP, unsigned int size, const int blocks, const int threads, int *hashKeys, int hashTableElements)
{

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_b, KernelP, size * sizeof(unsigned long long), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy input failed!");
    }

	//cudaStatus = hipMemset(dev_g, 0, size * hashTableElements * 4 * sizeof(int));
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy input failed!");
	//}

//	cudaStatus = hipMemset(dev_h, 0, size * hashTableElements * sizeof(unsigned long long));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy input failed!");
//	}


	//hipEvent_t start, stop;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);
    // Launch a kernel on the GPU with one thread for each element.
	//hipEventRecord(start);
	hipStream_t stream0;
	//hipStream_t stream1;
	hipStreamCreate(&stream0);
	//hipStreamCreate(&stream1);
    addKernel1<<<blocks,threads,0,stream0>>>(dev_a, dev_b, dev_c, dev_e, dev_f, dev_g);
	//addKernel1<<<blocks,threads,0,stream1>>>(dev_a, dev_b, dev_c, dev_e, dev_f, dev_g, dev_h);
	//hipEventRecord(stop);

	//hipEventSynchronize(stop);
	//float milliseconds = 0;
	//hipEventElapsedTime(&milliseconds, start, stop);
	//printf("Time taken: %f ms \n", milliseconds);

    // Check for any errors launching the kernel
    //cudaStatus = hipGetLastError();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    //}
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(NOut, dev_a, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy output failed!");
    }
    
    return cudaStatus;
}

